#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <fstream>
#include <math.h>
#include <vector>
#include <string>
#include "Header.h"

__device__ void EnergyConservationLaws(const double2& PS, const double2& PU, const double2& s_1, const double2& u_1, //
    double2& s2, double2& u2, double T_do, double x, double y, double dV)
{
    s2.x = s_1.x - (T_do / dV) * PS.x - T_do * s_1.x * u_1.y / y;
    if (s2.x <= 0)
    {
        printf("Problemsssss! x = %lf, y = %lf, ro = %lf, T = %lf, ro = %lf \n", x, y, s2.x, T_do, s_1.x);
        s2.x = s_1.x;
    }
    u2.x = (s_1.x * u_1.x - (T_do / dV) * PU.x - T_do * s_1.x * u_1.y * u_1.x / y) / s2.x;
    u2.y = (s_1.x * u_1.y - (T_do / dV) * PU.y - T_do * s_1.x * u_1.y * u_1.y / y) / s2.x;
    s2.y = (((s_1.y / (ggg - 1) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) - (T_do / dV) * PS.y - //
        T_do * u_1.y * (ggg * s_1.y / (ggg - 1) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) / y) - //
        0.5 * s2.x * (u2.x * u2.x + u2.y * u2.y)) * (ggg - 1);

    if (s2.y <= 0)
    {
        s2.y = 0.000001;
    }
}


void InitialConditions(double2& s2, double2& u2, double x, double y)
{
    // u2 - ��������, s2 - ��������� � ��������
    if (x * x + y * y < 10000)
    {
        s2.x = 8.0;
        s2.y = 2.0;
        u2.x = -90.0;
        u2.y = 0.0; 
    }
    else
    {
        s2.x = 1.0;
        s2.y = 1.0;
        u2.x = -1.0;
        u2.y = 0.0;
    }
   
}


void Initialization(int& N_, int& M_, int& step_1_, int& step_2_, int& step_3_, int& step_4_, int& step_5_, int& step_6_,//
    bool& ots__, double& x_min_, double& x_max_, double& y_max_, double& U_ots_, double& krit_, int& Nmin_, double& dist_1_)
{
    // ��������� ������� ����� ������:
    N_ = 256; // 7167 //1792 //1792                 // ���������� ����� �� x
    M_ = 256; // //1280 //1280                 // ���������� ����� �� y
    // �����:  N*M ������ ������� �� 256!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    x_min_ = -2760.0;
    x_max_ = 450.0;
    y_max_ = 2250.0;

    dist_1_ = 110.0;     // ���������� ���������� ����� (� ������� ���������� ����), ������ �� �� ���������
    Nmin_ = 5;              // ������ ����� ����� �������?

    ots__ = false;   // ������ ������� ������ �������� ����� ������ ������
    U_ots_ = -5.0;  // ����� ������ ����� ��������

    step_1_ = 5000;   // ���������� ����� �� ������� ������� HLL
    step_2_ = 0;   // ���������� ����� �� ������� ������� HLLC
    step_3_ = 0;   // ���������� ����� �� ������� ������� GODUNOV
    step_4_ = 0;   // ���������� ����� �� ������� ������� HLL + TVD
    step_5_ = 0;   // ���������� ����� �� ������� ������� HLLC + TVD
    step_6_ = 0;   // ���������� ����� �� ������� ������� GODUNOV + TVD

    krit_ = 0.3;    // �������� �������-���������-����  (<1)
}