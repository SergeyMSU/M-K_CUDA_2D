#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <fstream>
#include <math.h>
#include <vector>
#include <string>
#include "Header.h"
#include "sensor.h"
#include <omp.h>
#include <mutex>

using namespace std;

//__device__ int sign_(const double& x);
//__device__ double minmod_(double x, double y);
//__device__ double linear_(double x1, double t1, double x2, double t2, double x3, double t3, double y);
//__device__ void linear2_(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
//    double& A, double& B);



__device__ double minmod_(double x, double y)
{
    if (sign_(x) + sign_(y) == 0)
    {
        return 0.0;
    }
    else
    {
        return   ((sign_(x) + sign_(y)) / 2.0) * min(fabs(x), fabs(y));  ///minmod
        //return (2*x*y)/(x + y);   /// vanleer
    }
}

__device__ double linear_(double x1, double t1, double x2, double t2, double x3, double t3, double y)
{
    double d = minmod_((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    return  (d * (y - x2) + t2);
}

__device__ void linear2_(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
    double& A, double& B)
{
    // ������� �������� - ����������� - �� ������� �� ����
    double d = minmod_((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    A = (d * (y1 - x2) + t2);
    B = (d * (y2 - x2) + t2);
    //printf("%lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf \n", x1, t1, x2, t2, x3, t3, y1, y2, A, B);
    return;
}

__device__ int sign_(const double& x)
{
    if (x > 0)
    {
        return 1;
    }
    else if (x < 0)
    {
        return  -1;
    }
    else
    {
        return 0;
    }
}


__device__ void TVD(const double2& s_1, const double2& s_2, const double2& s_3, const double2& s_4, const double2& s_5,//
    const double2& s_6, const double2& s_7, const double2& s_8, const double2& s_9, double2& s12,//
    double2& s13, double2& s14, double2& s15, double2& s21, double2& s31, double2& s41, double2& s51, double ddx, double ddy, bool zero)
{
    // ��� ��������� � �������� zero ������ ���� ����� true
    linear2_(-ddx, s_4.x, 0.0, s_1.x, ddx, s_2.x, -ddx / 2.0, ddx / 2.0, s14.x, s12.x);
    if (zero == true)
    {
        if (s14.x <= 0.0)
        {
            s14.x = s_1.x;
        }
        if (s12.x <= 0.0)
        {
            s12.x = s_1.x;
        }
    }
   
    linear2_(-ddx, s_4.y, 0.0, s_1.y, ddx, s_2.y, -ddx / 2.0, ddx / 2.0, s14.y, s12.y);
    if (zero == true)
    {
        if (s14.y <= 0.0)
        {
            s14.y = s_1.y;
        }
        if (s12.y <= 0.0)
        {
            s12.y = s_1.y;
        }
    }

    linear2_(-ddy, s_3.x, 0.0, s_1.x, ddy, s_5.x, -ddy / 2.0, ddy / 2.0, s13.x, s15.x);
    if (zero == true)
    {
        if (s13.x <= 0.0)
        {
            s13.x = s_1.x;
        }
        if (s15.x <= 0.0)
        {
            s15.x = s_1.x;
        }
    }

    linear2_(-ddy, s_3.y, 0.0, s_1.y, ddx, s_5.y, -ddy / 2.0, ddy / 2.0, s13.y, s15.y);
    if (zero == true)
    {
        if (s13.y <= 0.0)
        {
            s13.y = s_1.y;
        }
        if (s15.y <= 0.0)
        {
            s15.y = s_1.y;
        }
    }

    s21.x = linear_(0.0, s_1.x, ddx, s_2.x, 2.0 * ddx, s_6.x, ddx / 2.0);
    if (s21.x <= 0) s21.x = s_2.x;
    s21.y = linear_(0.0, s_1.y, ddx, s_2.y, 2.0 * ddx, s_6.y, ddx / 2.0);
    if (s21.y <= 0) s21.y = s_2.y;

    s41.x = linear_(0.0, s_1.x, - ddx, s_4.x, - 2.0 * ddx, s_8.x, - ddx / 2.0);
    if (s41.x <= 0 && zero == true) s41.x = s_4.x;
    s41.y = linear_(0.0, s_1.y, - ddx, s_4.y, - 2.0 * ddx, s_8.y, - ddx / 2.0);
    if (s41.y <= 0 && zero == true) s41.y = s_4.y;

    s31.x = linear_(0.0, s_1.x, - ddy, s_3.x, - 2.0 * ddy, s_7.x, - ddy / 2.0);
    if (s31.x <= 0 && zero == true) s31.x = s_3.x;
    s31.y = linear_(0.0, s_1.y, - ddy, s_3.y, - 2.0 * ddy, s_7.y, - ddy / 2.0);
    if (s31.y <= 0 && zero == true) s31.y = s_3.y;

    s51.x = linear_(0.0, s_1.x, + ddy, s_5.x, + 2.0 * ddy, s_9.x, + ddy / 2.0);
    if (s51.x <= 0) s51.x = s_5.x;
    s51.y = linear_(0.0, s_1.y, + ddy, s_5.y, + 2.0 * ddy, s_9.y, + ddy / 2.0);
    if (s51.y <= 0)  s51.y = s_5.y;

    return;
}


__device__ double HLLC_Korolkov_2D(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double n1, const double n2, double2& Ps, double2& Pu, const double rad)
{
    double u_L, v_L;
    double u_R, v_R;

    double ro1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;

    double ro2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;

    double t1 = -n2;    // ����������� ������
    double t2 = n1;

    u_L = u1 * n1 + v1 * n2;
    v_L = u1 * t1 + v1 * t2;

    u_R = u2 * n1 + v2 * n2;
    v_R = u2 * t1 + v2 * t2;

    double cL = sqrt(ga * p1 / ro1);
    double cR = sqrt(ga * p2 / ro2);

    /*double SL = min((u_L - cL), (u_R - cR));
    double SR = max((u_L + cL), (u_R + cR));*/

    double SL = min(u_L, u_R) - max(cL, cR);
    double SR = max(u_L, u_R) + max(cL, cR);

   /* double SL = min(u_L, u_R) - max(cL, cR);
    double SR = max(u_L, u_R) + max(cL, cR);*/

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    if (SL >= 0.0)
    {
        Ps.x = ro1 * u_L;
        Ps.y = ( ga * p1/(g1) + 0.5 * ro1 * (kv(u1) + kv(v1)) ) * u_L;
        Pu.x = (ro1 * u_L * u_L + p1) * n1 + (ro1 * u_L * v_L) * t1;
        Pu.y = (ro1 * u_L * u_L + p1) * n2 + (ro1 * u_L * v_L) * t2;
        return time;
    }
    else if (SR <= 0.0)
    {
        Ps.x = ro2 * u_R;
        Ps.y = (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;
        Pu.x = (ro2 * u_R * u_R + p2) * n1 + (ro2 * u_R * v_R) * t1;
        Pu.y = (ro2 * u_R * u_R + p2) * n2 + (ro2 * u_R * v_R) * t2;
        return time;
    }
    else
    {
        double SM = ( (SR - u_R)*ro2 * u_R - (SL - u_L)*ro1*u_L - p2 + p1 )/( (SR - u_R)*ro2 - (SL - u_L)*ro1 );
        double pp = p1 + ro1 * (SL - u_L) * (SM - u_L);

        if (SM <= 0.0)
        {
            double rr = ro2 * (SR - u_R) / (SR - SM);
            double e = p2 / g1 + 0.5 * ro2 * (kv(u2) + kv(v2));
            double ee = ((SR - u_R) * e - p2 * u_R + pp * SM) / (SR - SM);
            Ps.x = SR * (rr - ro2) + ro2 * u_R;
            Ps.y = SR * (ee - e) + (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;

            double F1 = (ro2 * u_R * u_R + p2) + SR * (rr * SM - ro2 * u_R);
            double F2 = (ro2 * u_R * v_R) + SR * (rr * v_R - ro2 * v_R);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else if (SM >= 0.0)
        {
            double rr = ro1 * (SL - u_L) / (SL - SM);
            double e = p1 / g1 + 0.5 * ro1 * (kv(u1) + kv(v1));
            double ee = ( (SL - u_L)*e - p1*u_L + pp * SM )/(SL - SM);
            Ps.x = SL * (rr - ro1) + ro1 * u_L;
            Ps.y = SL * (ee - e) + (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
            double F1 = (ro1 * u_L * u_L + p1) + SL * (rr * SM - ro1 * u_L);
            double F2 = (ro1 * u_L * v_L ) + SL * (rr * v_L - ro1 * v_L);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else
        {
            printf("ERROR HLLC_KOROLKOV_2d   kod oshibki: 1jdt27453h\n");
            return time;
        }
    }
    return time;
}

__device__ double HLLCQ_Korolkov_2D(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double& LQ, const double& RQ, double n1, double n2, double2& Ps, double2& Pu, double& PQ, double rad)
{
    double u_L, v_L;
    double u_R, v_R;

    double ro1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;
    double Q_L = LQ;

    double ro2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;
    double Q_R = RQ;

    double t1 = -n2;    // ����������� ������
    double t2 = n1;

    u_L = u1 * n1 + v1 * n2;
    v_L = u1 * t1 + v1 * t2;

    u_R = u2 * n1 + v2 * n2;
    v_R = u2 * t1 + v2 * t2;

    double cL = sqrt(ga * p1 / ro1);
    double cR = sqrt(ga * p2 / ro2);

    double SL = min((u_L - cL), (u_R - cR));
    double SR = max((u_L + cL), (u_R + cR));

   /* double SL = min(u_L, u_R) - max(cL, cR);
    double SR = max(u_L, u_R) + max(cL, cR);*/

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL1 = ro1 * u_L * u_L + p1;
    double FL2 = ro1 * u_L * v_L;

    double FR1 = ro2 * u_R * u_R + p2;
    double FR2 = ro2 * u_R * v_R;

    if (SL >= 0.0)
    {
        PQ = Q_L * u_L;
        Ps.x = ro1 * u_L;
        Ps.y = (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
        Pu.x = (FL1) * n1 + (FL2) * t1;
        Pu.y = (FL1) * n2 + (FL2) * t2;
        return time;
    }
    else if (SR <= 0.0)
    {
        PQ = Q_R * u_R;
        Ps.x = ro2 * u_R;
        Ps.y = (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;
        Pu.x = (FR1) * n1 + (FR2) * t1;
        Pu.y = (FR1) * n2 + (FR2) * t2;
        return time;
    }
    else
    {
        double SM = ((SR - u_R) * ro2 * u_R - (SL - u_L) * ro1 * u_L - p2 + p1) / ((SR - u_R) * ro2 - (SL - u_L) * ro1);
        double pp = p1 + ro1 * (SL - u_L) * (SM - u_L);

        if (SM <= 0.0)
        {
            double rr = ro2 * (SR - u_R) / (SR - SM);
            double e = p2 / g1 + 0.5 * ro2 * (kv(u2) + kv(v2));
            double ee = ((SR - u_R) * e - p2 * u_R + pp * SM) / (SR - SM);
            PQ = SR * (rr * Q_R/ro2 - Q_R) + Q_R * u_R;
            Ps.x = SR * (rr - ro2) + ro2 * u_R;
            Ps.y = SR * (ee - e) + (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;

            double F1 = (ro2 * u_R * u_R + p2) + SR * (rr * SM - ro2 * u_R);
            double F2 = (ro2 * u_R * v_R) + SR * (rr * v_R - ro2 * v_R);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else if (SM >= 0.0)
        {
            double rr = ro1 * (SL - u_L) / (SL - SM);
            double e = p1 / g1 + 0.5 * ro1 * (kv(u1) + kv(v1));
            double ee = ((SL - u_L) * e - p1 * u_L + pp * SM) / (SL - SM);
            PQ = SL * (rr * Q_L / ro1 - Q_L) + Q_L * u_L;
            Ps.x = SL * (rr - ro1) + ro1 * u_L;
            Ps.y = SL * (ee - e) + (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
            double F1 = FL1 + SL * (rr * SM - ro1 * u_L);
            double F2 = FL2 + SL * (rr * v_L - ro1 * v_L);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else
        {
            printf("ERROR HLLC_KOROLKOV_2d   kod oshibki: 1jdt27453h\n");
            return time;
        }
    }
    return time;
}

__device__ double HLLCQ_Aleksashov(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double& LQ, const double& RQ, double n1, double n2, double2& Ps, double2& Pu, double& PQ, double rad)
{
    double n[3];
    n[0] = n1;
    n[1] = n2;
    n[2] = 0.0;
    //int id_bn = 1;
    //int n_state = 1;
    double FR[8], FL[8];
    double UL[8], UZ[8], UR[8];
    double UZL[8], UZR[8];

    double vL[3], vR[3], bL[3], bR[3];
    double vzL[3], vzR[3], bzL[3], bzR[3];
    double qv[3];
    double aco[3][3];

    double wv = 0.0;
    double r1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double w1 = 0.0;
    double p1 = Ls.y;
    double bx1 = 0.0;
    double by1 = 0.0;
    double bz1 = 0.0;
    double Q_L = LQ;


    double r2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double w2 = 0.0;
    double p2 = Rs.y;
    double bx2 = 0.0;
    double by2 = 0.0;
    double bz2 = 0.0;
    double Q_R = RQ;

    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;
    double abx = (bx2 + bx1) / 2.0;
    double aby = (by2 + by1) / 2.0;
    double abz = (bz2 + bz1) / 2.0;


    double bk = abx * n[0] + aby * n[1] + abz * n[2];
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double d = b2 - kv(bk);
    aco[0][0] = n[0];
    aco[1][0] = n[1];
    aco[2][0] = n[2];
    if (d > 0.000000001)
    {
        d = sqrt(d);
        aco[0][1] = (abx - bk * n[0]) / d;
        aco[1][1] = (aby - bk * n[1]) / d;
        aco[2][1] = (abz - bk * n[2]) / d;
        aco[0][2] = (aby * n[2] - abz * n[1]) / d;
        aco[1][2] = (abz * n[0] - abx * n[2]) / d;
        aco[2][2] = (abx * n[1] - aby * n[0]) / d;
    }
    else
    {
        double aix, aiy, aiz;
        if ((fabs(n[0]) < fabs(n[1])) && (fabs(n[0]) < fabs(n[2])))
        {
            aix = 1.0;
            aiy = 0.0;
            aiz = 0.0;
        }
        else if (fabs(n[1]) < fabs(n[2]))
        {
            aix = 0.0;
            aiy = 1.0;
            aiz = 0.0;
        }
        else
        {
            aix = 0.0;
            aiy = 0.0;
            aiz = 1.0;
        }

        double aik = aix * n[0] + aiy * n[1] + aiz * n[2];
        d = sqrt(1.0 - kv(aik));
        aco[0][1] = (aix - aik * n[0]) / d;
        aco[1][1] = (aiy - aik * n[1]) / d;
        aco[2][1] = (aiz - aik * n[2]) / d;
        aco[0][2] = (aiy * n[2] - aiz * n[1]) / d;
        aco[1][2] = (aiz * n[0] - aix * n[2]) / d;
        aco[2][2] = (aix * n[1] - aiy * n[0]) / d;
    }

    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }

    double aaL = bL[0] / sqrt(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = sqrt(ga * p1 / r1);
    double qp = sqrt(b21 + cL * (cL + 2.0 * aaL));
    double qm = sqrt(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / 2.0;
    double ptL = p1 + b2L / 2.0;

    double aaR = bR[0] / sqrt(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = sqrt(ga * p2 / r2);
    qp = sqrt(b22 + cR * (cR + 2.0 * aaR));
    qm = sqrt(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / 2.0;
    double ptR = p2 + b2R / 2.0;

    double aC = (aaL + aaR) / 2.0;
    double b2o = (b22 + b21) / 2.0;
    double cC = sqrt(ga * ap / ro);
    qp = sqrt(b2o + cC * (cC + 2.0 * aC));
    qm = sqrt(b2o + cC * (cC - 2.0 * aC));
    double cfC = (qp + qm) / 2.0;
    double vC1 = (vL[0] + vR[0]) / 2.0;

    double SL = min((vL[0] - cfL), (vR[0] - cfR));
    double SR = max((vL[0] + cfL), (vR[0] + cfR));

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);

    if (SR <= SL)
    {
        printf("231\n");
    }

    double SM00 = SM;
    double SR00 = SR;
    double SL00 = SL;
    double SM01, SR01, SL01;
    if ((SM00 >= SR00) || (SM00 <= SL00))
    {
        SL = min((vL[0] - cfL), (vR[0] - cfR));
        SR = max((vL[0] + cfL), (vR[0] + cfR));
        suR = SR - vR[0];
        suL = SL - vL[0];
        SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);
        SM01 = SM;
        SR01 = SR;
        SL01 = SL;
        if ((SM01 >= SR01) || (SM01 <= SL01))
        {
            printf("251\n");
        }
    }


    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / 2.0;
    double e2 = p2 / g1 + r2 * upt2 + b2R / 2.0;

    double FL0 = Q_L * vL[0];
    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    double FR0 = Q_R * vR[0];
    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;


    for (int ik = 0; ik < 3; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UL[ik + 5] = bL[ik];
        UR[ik + 1] = r2 * vR[ik];
        UR[ik + 5] = bR[ik];
    }

    for (int ik = 0; ik < 8; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }

    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = ptR + r2 * suR * (SM - vR[0]);
    double ptzL = ptL + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;
    bzR[0] = UZ[5];
    bzL[0] = UZ[5];

    vzR[1] = UZ[2] / UZ[0];
    vzR[2] = UZ[3] / UZ[0];
    vzL[1] = vzR[1];
    vzL[2] = vzR[2];

    vzR[1] = vR[1] + UZ[5] * (bR[1] - UZ[6]) / suR / r2;
    vzR[2] = vR[2] + UZ[5] * (bR[2] - UZ[7]) / suR / r2;
    vzL[1] = vL[1] + UZ[5] * (bL[1] - UZ[6]) / suL / r1;
    vzL[2] = vL[2] + UZ[5] * (bL[2] - UZ[7]) / suL / r1;

    bzR[1] = UZ[6];
    bzR[2] = UZ[7];
    bzL[1] = bzR[1];
    bzL[2] = bzR[2];

    double sbvz = (UZ[5] * UZ[1] + UZ[6] * UZ[2] + UZ[7] * UZ[3]) / UZ[0];

    double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + UZ[5] * (sbv2 - sbvz)) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + UZ[5] * (sbv1 - sbvz)) / (SL - SM);

    if (fabs(UZ[5]) < 0.000000001)
    {
        vzR[1] = vR[1];
        vzR[2] = vR[2];
        vzL[1] = vL[1];
        vzL[2] = vL[2];
        bzR[1] = bR[1] * suRm;
        bzR[2] = bR[2] * suRm;
        bzL[1] = bL[1] * suLm;
        bzL[2] = bL[2] * suLm;
    }
    UZL[0] = rzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[4] = ezR;

    for (int ik = 0; ik < 3; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZL[ik + 5] = bzL[ik];
        UZR[ik + 1] = vzR[ik] * rzR;
        UZR[ik + 5] = bzR[ik];
    }

    if (SL > wv)
    {
        PQ = FL0;
        Ps.x = FL[0] - wv * UL[0];
        Ps.y = FL[4] - wv * UL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] - wv * UL[ik];
        }
    }
    else if ((SL <= wv) && (SM >= wv))
    {
        PQ = FL0 + SL * (rzL * Q_L/r1 - Q_L);
        Ps.x = FL[0] + SL * (rzL - r1) - wv * UZL[0];
        Ps.y = FL[4] + SL * (ezL - e1) - wv * UZL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
        }
    }
    else if ((SM <= wv) && (SR >= wv))
    {
        PQ = FR0 + SR * (rzR * Q_R/r2 - Q_R);
        Ps.x = FR[0] + SR * (rzR - r2) - wv * UZR[0];
        Ps.y = FR[4] + SR * (ezR - e2) - wv * UZR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
        }
    }
    else if (SR < wv)
    {
        PQ = FR0;
        Ps.x = FR[0] - wv * UR[0];
        Ps.y = FR[4] - wv * UR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + -wv * UR[ik];
        }
    }
    else
    {
        printf("DDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDD\n");
    }


    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1] + aco[0][2] * qv[2];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1] + aco[1][2] * qv[2];

    return time;
}

__device__ double HLLC_Aleksashov_2D(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double n[2];
    n[0] = n1;
    n[1] = n2;
    //int id_bn = 1;
    //int n_state = 1;
    double FR[5], FL[5];
    double UL[5], UZ[5], UR[5];
    double UZL[5], UZR[5];

    double vL[2], vR[2];
    double vzL[2], vzR[2];
    double qv[2];
    double aco[2][2];

    double r1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;


    double r2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;

    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;


    aco[0][0] = n[0];
    aco[1][0] = n[1];

    aco[0][1] = -n[1];
    aco[1][1] = n[0];
    

    for (int i = 0; i < 2; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2;
    }


    double cL = sqrt(ga * p1 / r1);
    double cR = sqrt(ga * p2 / r2);


    double cC = sqrt(ga * ap / ro);

    double vC1 = (vL[0] + vR[0]) / 2.0;

    //double SL = min((vL[0] - cL), (vR[0] - cR));
    //double SR = max((vL[0] + cL), (vR[0] + cR));

    double SL = min(vL[0], vR[0]) - max(cL, cR);
    double SR = max(vL[0], vR[0]) + max(cL, cR);


    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - p2 + p1 - suL * r1 * vL[0]) / (suR * r2 - suL * r1);


    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double upt1 = (kv(u1) + kv(v1)) / 2.0;

    double upt2 = (kv(u2) + kv(v2)) / 2.0;

    double e1 = p1 / g1 + r1 * upt1;
    double e2 = p2 / g1 + r2 * upt2;

    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + p1;
    FL[2] = r1 * vL[0] * vL[1];
    FL[4] = (e1 + p1) * vL[0];

    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + p2;
    FR[2] = r2 * vR[0] * vR[1];
    FR[4] = (e2 + p2) * vR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;


    for (int ik = 0; ik < 2; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UR[ik + 1] = r2 * vR[ik];
    }

    for (int ik = 0; ik < 5; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }

    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = p2 + r2 * suR * (SM - vR[0]);
    double ptzL = p1 + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;


    vzR[1] = vR[1];
    vzL[1] = vL[1];



    double ezR = e2 * suRm + (ptz * SM - p2 * vR[0]) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - p1 * vL[0]) / (SL - SM);

    UZL[0] = rzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[4] = ezR;

    for (int ik = 0; ik < 2; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZR[ik + 1] = vzR[ik] * rzR;
    }

    if (SL > 0.0)
    {
        Ps.x = FL[0];
        Ps.y = FL[4];
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FL[ik];
        }
    }
    else if ((SL <= 0.0) && (SM >= 0.0))
    {
        Ps.x = FL[0] + SL * (rzL - r1);
        Ps.y = FL[4] + SL * (ezL - e1);
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]);
        }
    }
    else if ((SM <= 0.0) && (SR >= 0.0))
    {
        Ps.x = FR[0] + SR * (rzR - r2);
        Ps.y = FR[4] + SR * (ezR - e2);
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]);
        }
    }
    else if (SR < 0.0)
    {
        Ps.x = FR[0];
        Ps.y = FR[4];
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FR[ik];
        }
    }
    else
    {
        printf("hllc 2d ERROR\n");
    }


    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1];

    return time;
}

__device__ int sign(double& x)
{
    if (x > 0)
    {
        return 1;
    }
    else if (x < 0)
    {
        return  -1;
    }
    else
    {
        return 0;
    }
}

__device__ double HLLDQ_Korolkov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, double& rad, int metod)
{// �� ��������, ���� �������� ����� �� �������
 // ������� ����� ��������� �� ���� ���������

    double bx_L = Bx_L / spi4;
    double by_L = By_L / spi4;
    double bz_L = Bz_L / spi4;

    double bx_R = Bx_R / spi4;
    double by_R = By_R / spi4;
    double bz_R = Bz_R / spi4;

    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;

    double m1 = 0.0;
    double m2 = 0.0;
    double m3 = 0.0;

    if (n1 > 0.1)
    {
        t2 = 1.0;
        m3 = 1.0;
    }
    else if (n2 > 0.1)
    {
        t3 = 1.0;
        m1 = 1.0;
    }
    else if (n3 > 0.1)
    {
        t1 = 1.0;
        m2 = 1.0;
    }
    else if (n1 < -0.1)
    {
        t3 = -1.0;
        m2 = -1.0;
    }
    else if (n2 < -0.1)
    {
        t1 = -1.0;
        m3 = -1.0;
    }
    else if (n3 < -0.1)
    {
        t1 = -1.0;
        m2 = -1.0;
    }
    else
    {
        printf("EROROR 1421  normal_error\n");
    }


    double u1, v1, w1, u2, v2, w2;
    u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
    v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
    w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;
    u2 = v1_R * n1 + v2_R * n2 + v3_R * n3;
    v2 = v1_R * t1 + v2_R * t2 + v3_R * t3;
    w2 = v1_R * m1 + v2_R * m2 + v3_R * m3;

    double bn1, bt1, bm1, bn2, bt2, bm2;
    bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
    bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
    bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
    bn2 = bx_R * n1 + by_R * n2 + bz_R * n3;
    bt2 = bx_R * t1 + by_R * t2 + bz_R * t3;
    bm2 = bx_R * m1 + by_R * m2 + bz_R * m3;

    //cout << " = " << bt2 * bt2 + bm2 * bm2 << endl;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double ca_L = bn1 / sqrtroL;
    double ca_R = bn2 / sqrtroR;
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);

    double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
    double bb_R = kv(bx_R) + kv(by_R) + kv(bz_R);

    double aL = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;
    double aR = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;

    double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R) + kv(v3_R)) / 2.0;

    double cfL = sqrt((ggg * p_L + bb_L + //
        sqrt(kv(ggg * p_L + bb_L) - 4.0 * ggg * p_L * kv(bn1))) / (2.0 * ro_L));
    double cfR = sqrt((ggg * p_R + bb_R + //
        sqrt(kv(ggg * p_R + bb_R) - 4.0 * ggg * p_R * kv(bn2))) / (2.0 * ro_R));


    double SL = min(u1, u2) - max(cfL, cfR);
    double SR = max(u1, u2) + max(cfL, cfR);

    double pTL = p_L + bb_L / 2.0;
    double pTR = p_R + bb_R / 2.0;

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - pTR + pTL) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * pTL - suL * ro_L * pTR + ro_L * ro_R * suR * suL * (u2 - u1))//
        / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL[9], FR[9], UL[9], UR[9];

    double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;
    double e2 = p_R / g1 + ro_R * uu_R + bb_R / 2.0;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
    FL[2] = ro_L * u1 * v1 - bn1 * bt1;
    FL[3] = ro_L * u1 * w1 - bn1 * bm1;
    FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
    //cout << uu_L << endl;
    FL[5] = 0.0;
    FL[6] = u1 * bt1 - v1 * bn1;
    FL[7] = u1 * bm1 - w1 * bn1;
    FL[8] = Q_L * u1;

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + pTR - kv(bn2);
    FR[2] = ro_R * u2 * v2 - bn2 * bt2;
    FR[3] = ro_R * u2 * w2 - bn2 * bm2;
    FR[4] = (e2 + pTR) * u2 - bn2 * (u2 * bn2 + v2 * bt2 + w2 * bm2);
    FR[5] = 0.0;
    FR[6] = u2 * bt2 - v2 * bn2;
    FR[7] = u2 * bm2 - w2 * bn2;
    FR[8] = Q_R * u2;

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = ro_L * w1;
    UL[4] = e1;
    UL[5] = bn1;
    UL[6] = bt1;
    UL[7] = bm1;
    UL[8] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = ro_R * w2;
    UR[4] = e2;
    UR[5] = bn2;
    UR[6] = bt2;
    UR[7] = bm2;
    UR[8] = Q_R;

    double bn = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
    double bt = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
    double bm = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
    double bbn = bn * bn;

    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);

    if (metod == 2)   // HLLC  + mgd
    {
        double sbv1 = u1 * bn1 + v1 * bt1 + w1 * bm1;
        double sbv2 = u2 * bn2 + v2 * bt2 + w2 * bm2;

        double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
        double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
        double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
        double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
        double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
        double vzL, vzR, vLL, wLL, vRR, wRR, ppLR, btt1, bmm1, btt2, bmm2, ee1, ee2;


        double suRm = suR / (SR - SM);
        double suLm = suL / (SL - SM);
        double rzR = ro_R * suRm;
        double rzL = ro_L * suLm;

        double ptzR = pTR + ro_R * suR * (SM - u2);
        double ptzL = pTL + ro_L * suL * (SM - u1);
        double ptz = (ptzR + ptzL) / 2.0;


        vRR = UZ2 / UZ0;
        wRR = UZ3 / UZ0;
        vLL = vRR;
        wLL = wRR;

        /*vRR = v2 + bn * (bt2 - bt) / suR / ro_R;
        wRR = w2 + bn * (bm2 - bm) / suR / ro_R;
        vLL = v1 + bn * (bt1 - bt) / suL / ro_L;
        wLL = w1 + bn * (bm1 - bm) / suL / ro_L;*/

        btt2 = bt;
        bmm2 = bm;
        btt1 = btt2;
        bmm1 = bmm2;

        double sbvz = (bn * UZ1 + bt * UZ2 + bm * UZ3) / UZ0;

        ee2 = e2 * suRm + (ptz * SM - pTR * u2 + bn * (sbv2 - sbvz)) / (SR - SM);
        ee1 = e1 * suLm + (ptz * SM - pTL * u1 + bn * (sbv1 - sbvz)) / (SL - SM);

        /*if (fabs(bn) < 0.000001 )
        {
            vRR = v2;
            wRR = w2;
            vLL = v1;
            wLL = w1;
            btt2 = bt2 * suRm;
            bmm2 = bm2 * suRm;
            btt1 = bt1 * suLm;
            bmm1 = bm1 * suLm;
        }*/

        /*ppLR = (pTL + ro_L * (SL - u1) * (SM - u1) + pTR + ro_R * (SR - u2) * (SM - u2)) / 2.0;

        if (fabs(bn) < 0.000001)
        {
            vLL = v1;
            wLL = w1;
            vRR = v2;
            wRR = w2;

            btt1 = bt1 * (SL - u1) / (SL - SM);
            btt2 = bt2 * (SR - u2) / (SR - SM);

            bmm1 = bm1 * (SL - u1) / (SL - SM);
            bmm2 = bm2 * (SR - u2) / (SR - SM);

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTL * u2 + ppLR * SM) / (SR - SM);
        }
        else
        {
            btt2 = btt1 = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
            bmm2 = bmm1 = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
            vLL = v1 + bn * (bt1 - btt1) / (ro_L * (SL - u1));
            vRR = v2 + bn * (bt2 - btt2) / (ro_R * (SR - u2));

            wLL = w1 + bn * (bm1 - bmm1) / (ro_L * (SL - u1));
            wRR = w2 + bn * (bm2 - bmm2) / (ro_R * (SR - u2));

            double sks1 = u1 * bn1 + v1 * bt1 + w1 * bm1 - SM * bn - vLL * btt1 - wLL * bmm1;
            double sks2 = u2 * bn2 + v2 * bt2 + w2 * bm2 - SM * bn - vRR * btt2 - wRR * bmm2;

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM + bn * sks1) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTR * u2 + ppLR * SM + bn * sks2) / (SR - SM);
        }*/


        double  ULL[9], URR[9], PO[9];
        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = ee1;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = ee2;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        if (SL >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SM >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
        }
        else if (SR > 0.0 && SM < 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
        }
        else if (SR <= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;

    }
    else if (metod == 3)  // HLLD
    {

        double ttL = ro_L * suL * (SL - SM) - bbn;
        double ttR = ro_R * suR * (SR - SM) - bbn;

        double vLL, wLL, vRR, wRR, btt1, bmm1, btt2, bmm2;

        if (fabs(ttL) >= 0.000001)
        {
            vLL = v1 - bn * bt1 * (SM - u1) / ttL;
            wLL = w1 - bn * bm1 * (SM - u1) / ttL;
            btt1 = bt1 * (ro_L * suL * suL - bbn) / ttL;
            bmm1 = bm1 * (ro_L * suL * suL - bbn) / ttL;
        }
        else
        {
            vLL = v1;
            wLL = w1;
            btt1 = 0.0;
            bmm1 = 0.0;
        }

        if (fabs(ttR) >= 0.000001)
        {
            vRR = v2 - bn * bt2 * (SM - u2) / ttR;
            wRR = w2 - bn * bm2 * (SM - u2) / ttR;
            btt2 = bt2 * (ro_R * suR * suR - bbn) / ttR;
            bmm2 = bm2 * (ro_R * suR * suR - bbn) / ttR;
            //cout << "tbr = " << (ro_R * suR * suR - bbn) / ttR << endl;
            //cout << "bt2 = " << bt2 << endl;
        }
        else
        {
            vRR = v2;
            wRR = w2;
            btt2 = 0.0;
            bmm2 = 0.0;
        }

        double eLL = (e1 * suL + PTT * SM - pTL * u1 + bn * //
            ((u1 * bn1 + v1 * bt1 + w1 * bm1) - (SM * bn + vLL * btt1 + wLL * bmm1))) //
            / (SL - SM);
        double eRR = (e2 * suR + PTT * SM - pTR * u2 + bn * //
            ((u2 * bn2 + v2 * bt2 + w2 * bm2) - (SM * bn + vRR * btt2 + wRR * bmm2))) //
            / (SR - SM);

        double sqrtroLL = sqrt(ro_LL);
        double sqrtroRR = sqrt(ro_RR);
        double SLL = SM - fabs(bn) / sqrtroLL;
        double SRR = SM + fabs(bn) / sqrtroRR;

        double idbn = 1.0;
        if (fabs(bn) > 0.001)
        {
            idbn = 1.0 * sign(bn);
        }
        else
        {
            idbn = 0.0;
            SLL = SM;
            SRR = SM;
        }

        double vLLL = (sqrtroLL * vLL + sqrtroRR * vRR + //
            idbn * (btt2 - btt1)) / (sqrtroLL + sqrtroRR);

        double wLLL = (sqrtroLL * wLL + sqrtroRR * wRR + //
            idbn * (bmm2 - bmm1)) / (sqrtroLL + sqrtroRR);

        double bttt = (sqrtroLL * btt2 + sqrtroRR * btt1 + //
            idbn * sqrtroLL * sqrtroRR * (vRR - vLL)) / (sqrtroLL + sqrtroRR);

        double bmmm = (sqrtroLL * bmm2 + sqrtroRR * bmm1 + //
            idbn * sqrtroLL * sqrtroRR * (wRR - wLL)) / (sqrtroLL + sqrtroRR);

        double eLLL = eLL - idbn * sqrtroLL * ((SM * bn + vLL * btt1 + wLL * bmm1) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        double eRRR = eRR + idbn * sqrtroRR * ((SM * bn + vRR * btt2 + wRR * bmm2) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        //cout << " = " << bn << " " << btt2 << " " << bmm2 << endl;
        //cout << "sbvr = " << (SM * bn + vRR * btt2 + wRR * bmm2) << endl;
        double  ULL[9], URR[9], ULLL[9], URRR[9];

        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = eLL;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        //cout << ro_RR << endl;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = eRR;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        ULLL[0] = ro_LL;
        ULLL[1] = ro_LL * SM;
        ULLL[2] = ro_LL * vLLL;
        ULLL[3] = ro_LL * wLLL;
        ULLL[4] = eLLL;
        ULLL[5] = bn;
        ULLL[6] = bttt;
        ULLL[7] = bmmm;
        ULLL[8] = Q_LL;

        URRR[0] = ro_RR;
        URRR[1] = ro_RR * SM;
        URRR[2] = ro_RR * vLLL;
        URRR[3] = ro_RR * wLLL;
        URRR[4] = eRRR;
        URRR[5] = bn;
        URRR[6] = bttt;
        URRR[7] = bmmm;
        URRR[8] = Q_RR;

        double PO[9];

        if (SL >= 0.0)
        {
            //cout << "SL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SLL >= 0.0)
        {
            //cout << "SL < 0.0 && SLL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
            //cout << ULL[0] << endl;
        }
        else if (SLL <= 0.0 && SM >= 0.0)
        {
            //cout << "SLL <= 0.0 && SM >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SLL * ULLL[i] - (SLL - SL) * ULL[i] - SL * UL[i];
            }
        }
        else if (SM < 0.0 && SRR > 0.0)
        {
            //cout << "SM < 0.0 && SRR > 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SRR * URRR[i] - (SRR - SR) * URR[i] - SR * UR[i];
            }
            //cout << "P4 = " << URRR[4] << endl;
        }
        else if (SR > 0.0 && SRR <= 0.0)
        {
            //cout << "SR > 0.0 && SRR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
            //cout << URR[0] << endl;
        }
        else if (SR <= 0.0)
        {
            //cout << "SR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }

}

void spherical_skorost(double x, double y, double z, double Vx, double Vy, double Vz, double& Vr, double& Vphi, double& Vtheta)
{
    double r_1 = sqrt(x * x + y * y + z * z);
    double the_1 = acos(z / r_1);
    double phi_1 = polar_angle(x, y);

    Vr = Vx * sin(the_1) * cos(phi_1) + Vy * sin(the_1) * sin(phi_1) + Vz * cos(the_1);
    Vtheta = Vx * cos(the_1) * cos(phi_1) + Vy * cos(the_1) * sin(phi_1) - Vz * sin(the_1);
    Vphi = -Vx * sin(phi_1) + Vy * cos(phi_1);
}

void dekard_skorost(double x, double y, double z, double Vr, double Vphi, double Vtheta, double& Vx, double& Vy, double& Vz)
{
    double r_2 = sqrt(x * x + y * y + z * z);
    double the_2 = acos(z / r_2);
    double phi_2 = polar_angle(x, y);

    Vx = Vr * sin(the_2) * cos(phi_2) + Vtheta * cos(the_2) * cos(phi_2) - Vphi * sin(phi_2);
    Vy = Vr * sin(the_2) * sin(phi_2) + Vtheta * cos(the_2) * sin(phi_2) + Vphi * cos(phi_2);
    Vz = Vr * cos(the_2) - Vtheta * sin(the_2);
    
}

double polar_angle(double x, double y)
{
    if (x < 0)
    {
        return atan(y / x) + 1.0 * PI;
    }
    else if (x > 0 && y >= 0)
    {
        return atan(y / x);
    }
    else if (x > 0 && y < 0)
    {
        return atan(y / x) + 2.0 * PI;
    }
    else if (y > 0 && x >= 0 && x <= 0)
    {
        return PI / 2.0;
    }
    else if (y < 0 && x >= 0 && x <= 0)
    {
        return  3.0 * PI / 2.0;
    }
    return 0.0;
}

void M_K(vector<Sensor*> Sensors, const double2* s, const double2* u, double* nn1, double3* nn2, double* nn3)
{
    mutex mut_1;
    mutex* mut;
    mut = new mutex[K];

#pragma omp parallel for
    for (int index = 0; index < 270; index++)
    {
        bool info = false;
        if (index == 0)
        {
            info = true;
        }
        double ksi1, ksi2, x_0, y_0, z_0, r_0, ksi3, ksi4, ksi5, ksi6, ksi7, phi, Vr, Vphi, Vx;
        bool t = false;
        double mu1, mu2, mu3, mu4;
        int n, m;
        //mu1 = 1.0;
        mu1 = ((sqv_1) / sum_s) *(1.0 * AllNumber / Number1);
        mu2 = ((sqv_2) / sum_s) * (1.0 * AllNumber / Number2);
        mu3 = ((sqv_3) / sum_s) * (1.0 * AllNumber / Number3);
        mu4 = ((sqv_4) / sum_s) * (1.0 * AllNumber / Number4);
        double k;
        Sensor* sens = Sensors[index];
        mut_1.lock();
        cout << index << " potok  is  270" << endl;
        mut_1.unlock();

        for (int ii = 0; ii < Number1/270; ii++)
        {
            //cout << ii << endl;
            t = false;
            double a, b, c;
            Velosity_initial(sens, a, b, c);
            ksi1 = sens->MakeRandom();
            ksi2 = sens->MakeRandom();
            
            r_0 = sqrt(1.0 + ksi1 * (kv(y_max) - 1.0));
            phi = ksi2 * 2.0 * pi;
            y_0 = r_0 * cos(phi);
            z_0 = r_0 * sin(phi);

            //cout << a << endl;

            
            Belong_point(x_max + dx/2.0 - geo, r_0, n, m);  // ������� ������, ������� ����������� �����

            Fly_exchenge(sens, x_max + dx/2.0 - geo, y_0, z_0,//
                a, b, c, //
                m * N + n, s, u, mu1, nn1, nn2, nn3, ii, mut, info);
            //cout << "Stop 1" << endl;
        }
        for (int ii = 0; ii < Number2 / 270; ii++)  // � ������� �����������
        {
            t = false;
            //double a, b, c;
            //Velosity_initial(sens, a, b, c);
            ksi1 = sens->MakeRandom();
            ksi2 = sens->MakeRandom();
            ksi3 = sens->MakeRandom();
            ksi4 = sens->MakeRandom();
            ksi5 = sens->MakeRandom();
            ksi6 = sens->MakeRandom();
            ksi7 = sens->MakeRandom();

            x_0 = (x_min - dx / 2.0 + geo) + ksi1 * (x_max + dx - 2.0 * geo - x_min);
            phi = ksi2 * 2.0 * pi;
            Vphi = cos(2.0 * pi * ksi3) * sqrt(-log(1.0 - ksi4));
            Vx = Velosity_inf +  sin(2.0 * pi * ksi5) * sqrt(-log(1.0 - ksi6));
            Vr = -sqrt(-log(ksi7));
            y_0 = (y_max - geo) * cos(phi);
            z_0 = (y_max - geo) * sin(phi);


            Belong_point(x_0, y_max - geo, n, m);  // ������� ������, ������� ����������� �����

            Fly_exchenge(sens, x_0, y_0, z_0, Vx, cos(phi) * Vr - sin(phi)*Vphi,//
                sin(phi) * Vr + cos(phi) * Vphi, m * N + n, s, u, mu2, nn1, nn2, nn3, ii, mut, info);
        }
        for (int ii = 0; ii < Number3/270; ii++)
        {
            //cout << ii << endl;
            t = false;
            double a, b, c;
            Velosity_initial2(sens, a, b, c);
            ksi1 = sens->MakeRandom();
            ksi2 = sens->MakeRandom();

            r_0 = sqrt(ksi1 * y_max * y_max);
            phi = ksi2 * 2.0 * pi;
            y_0 = r_0 * cos(phi);
            z_0 = r_0 * sin(phi);




            Belong_point(x_min - dx / 2.0 + geo, r_0, n, m);  // ������� ������, ������� ����������� �����

            Fly_exchenge(sens, x_min - dx / 2.0 + geo, y_0, z_0,//
                a, b, c, //
                m * N + n, s, u, mu3, nn1, nn2, nn3, ii, mut, info);
        }
        for (int ii = 0; ii < Number4 / 270; ii++)
        {
            //cout << ii << endl;
            t = false;
            double a, b, c;
            Velosity_initial(sens, a, b, c);
            ksi1 = sens->MakeRandom();
            ksi2 = sens->MakeRandom();

            r_0 = sqrt(ksi1 * 1.0 * 1.0);
            phi = ksi2 * 2.0 * pi;
            y_0 = r_0 * cos(phi);
            z_0 = r_0 * sin(phi);

            //cout << a << endl;


            Belong_point(x_max + dx / 2.0 - geo, r_0, n, m);  // ������� ������, ������� ����������� �����

            Fly_exchenge(sens, x_max + dx / 2.0 - geo, y_0, z_0,//
                a, b, c, //
                m * N + n, s, u, mu4, nn1, nn2, nn3, ii, mut, info);
            //cout << "Stop 1" << endl;
        }
    }

    for (int k = 0; k < K; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        double no = (1.0 * AllNumber * (pi * kv(y + dy / 2.0) * dx - pi * kv(y - dy / 2.0) * dx));
        nn1[k] = sum_s * nn1[k] / no;
        nn2[k].x = sum_s * nn2[k].x / no;
        nn2[k].y = sum_s * nn2[k].y / no;
        nn2[k].z = sum_s * nn2[k].z / no;
        nn3[k] = sum_s * nn3[k] / no;
    }
}

void Fly_exchenge(Sensor* sens, double x_0, double y_0, double z_0, double Vx, double Vy, double Vz, int ind, //
    const double2* s, const double2* u, double mu, double* nn1, double3* nn2, double* nn3, int num, mutex* mut, bool info)
{
    int next = -1;
    int prev = -1;
    int head = ind;
    double X = x_0, Y = y_0, Z = z_0;
    double KSI = -log(1.0 - sens->MakeRandom());
    double I_do = 0.0;
    bool error = false;
    int per = 0;

    double Ur, Uphi, Utheta;
    double Vr, Vphi, Vtheta;
    double uu, vv, ww;

    do
    {
        if (Flying_exchange(KSI, Vx, Vy, Vz, X, Y, Z, next, head, prev, mu, I_do, s[head].x, u[head].x, u[head].y,//
            nn1, nn2, nn3, error, mut[head]) == false)
        {
            if (error == true)
            {
                watch(num);
            }
            break;
        }
        if (KSI < 0.0)
        {
            per = 1;
            KSI = -log(1.0 - sens->MakeRandom());
            I_do = 0.0;
            prev = head;
            double sk = sqrt(s[head].y / s[head].x);
            double alpha = polar_angle(Y, Z);
            double uuu, vvv;
            uuu = u[head].x;
            vvv = u[head].y;

            Change_Velosity(sens, uuu / sk, vvv * cos(alpha) / sk, vvv * sin(alpha) / sk,//
                             Vx / sk, Vy / sk, Vz / sk, uu, vv, ww);  // ����� ��������  r, theta, phi
            Vx = uu * sk;
            Vy = vv * sk;
            Vz = ww * sk;
        }
        else
        {
            per = 0;
            prev = head;
            head = next;
        }
    } while (true);

    return;
}

bool Flying_exchange(double& KSI, double& Vx, double& Vy, double& Vz, double& X, double& Y,//
    double& Z, int& next, int head, int prev, const double& mu, double& I_do, const double& ro, //
    const double& vx, const double& vy, double* nn1, double3* nn2, double* nn3, bool& error, mutex& mut)
    // Vx, Vy, Vz - �������� ����� ��������
    // X,Y,Z - ���������� �����
    // head - ����� ������� ������
{
    int n = head % N;                                   // ����� ������ �� x (�� 0)
    int m = (head - n) / N;                             // ����� ������ �� y (�� 0)
    //cout << n << " " << m << endl;
    double y0 = y_min + m * dy;
    double x0 = x_min + n * dx;
    double t1, t2, t3, x, uz;// y, z, r;
    //watch(ro);
    //cout << X << " " << Y << " " << Z << " " << Vx << " " << Vy << " " << Vz << endl;
    //cout << x0 << " " << y0 << endl;
    int mode = 0;
    double time = 1000000000;
    int step = 0;

    while(Peresechenie(x0, y0, X, Y, Z, Vx, Vy, Vz, mode, time) == false)
    {
        step++;
        if (step > 6)
        {
            cout << "Error  1605" << endl;
            return false;
        }
        double alpha = polar_angle(Y, Z);
        double yy, zz;
        yy = y0 * cos(alpha);
        zz = y0 * sin(alpha);
        double nn = sqrt(kv(X - x0) + kv(Y - yy) + kv(Z - zz));
        X = X -  geo * (X - x0)/nn;
        Y = Y -  geo * (Y - yy)/nn;
        Z = Z -  geo * (Z - zz)/nn;
    }


 
    if (mode == 1) // ���� �� ��������� � ������ ������, �� ���� ������ �� ����
    {
        t1 = time;
        /*x = X + t1 * Vx;
        y = Y + t1 * Vy;
        z = Z + t1 * Vz;*/
        double l = sqrt(kvv(t1 * Vx, t1 * Vy, t1 * Vz));
        double alpha = polar_angle(Y, Z);
        double x = sqrt(kvv(Vx - vx, Vy - vy * cos(alpha), Vz - vy * sin(alpha)));
        uz = exp(-x * x) / sqrtpi + (x + 1.0 / (2.0 * x)) * erf(x);
        double sig = Kn * sqrt(kvv(Vx,Vy,Vz)) / (ro * uz * sigma(uz));
        double I = I_do + l / sig;
        if (I < KSI || ChEx == false)  // �� ��������� �����������
        {
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;
            I_do = I;
            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

           

            next = m * N + n + 1;

            if (n + 1 >= N)
            {
                return false;
            }
            else
            {
                return true;
            }
        }
        else  // ����������� ����
        {
            double ksi = (KSI - I_do) * sig;
            t1 = ksi / sqrt(kvv(Vx, Vy, Vz));
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;
            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = head;
            I_do = 0.0;
            KSI = -1.0;
            return true; 
        }
    }
    else if (mode == 2) // ���� �� ��������� � ����� ������, �� ���� ������ �� ����
    {
        t1 = time;
        /*x = X + t1 * Vx;
        y = Y + t1 * Vy;
        z = Z + t1 * Vz;*/
        double l = sqrt(kvv(t1 * Vx, t1 * Vy, t1 * Vz));
        double alpha = polar_angle(Y, Z);
        double x = sqrt(kvv(Vx - vx, Vy - vy * cos(alpha), Vz - vy * sin(alpha)));
        uz = exp(-x * x) / sqrtpi + (x + 1.0 / (2.0 * x)) * erf(x);
        double sig = Kn * sqrt(kvv(Vx, Vy, Vz)) / (ro * uz * sigma(uz));
        double I = I_do + l / sig;
        if (I < KSI || ChEx == false)  // �� ��������� �����������
        {
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;
            I_do = I;

            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = m * N + n - 1;

            if (n - 1 < 0)
            {
                return false;
            }
            else
            {
                return true;
            }
        }
        else  // ����������� ����
        {
            double ksi = (KSI - I_do) * sig;
            t1 = ksi / sqrt(kvv(Vx, Vy, Vz));
            if (t1 > time)
            {
                cout << "ERROR   2872t42o4t28743234" << endl;
            }
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;

            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = head;
            I_do = 0.0;
            KSI = -1.0;
            return true;
                        

        }
            
        
    }
    else if (mode == 3)
    {
        t1 = time;
        /*x = X + t1 * Vx;
        y = Y + t1 * Vy;
        z = Z + t1 * Vz;*/
        double l = sqrt(kvv(t1 * Vx, t1 * Vy, t1 * Vz));
        double alpha = polar_angle(Y, Z);
        double x = sqrt(kvv(Vx - vx, Vy - vy * cos(alpha), Vz - vy * sin(alpha)));
        uz = exp(-x * x) / sqrtpi + (x + 1.0 / (2.0 * x)) * erf(x);
        double sig = Kn * sqrt(kvv(Vx, Vy, Vz)) / (ro * uz * sigma(uz));
        double I = I_do + l / sig;
        if (I < KSI || ChEx == false)  // �� ��������� �����������
        {
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;
            I_do = I;

            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = (m + 1) * N + n;

            if (m + 1 >= M)
            {
                return false;
            }
            else
            {
                return true;
            }
        }
        else  // ����������� ����
        {
            double ksi = (KSI - I_do) * sig;
            t1 = ksi / sqrt(kvv(Vx, Vy, Vz));
            if (t1 > time)
            {
                cout << "ERROR   2872t42o4t28743234" << endl;
            }
            X = X + t1 * Vx;
            Y = Y + t1 * Vy;
            Z = Z + t1 * Vz;

            double alpha2 = polar_angle(Y - 0.5 * t1 * Vy, Z - 0.5 * t1 * Vz);
            
            mut.lock();
            nn1[head] += t1 * mu;
            //nn2[head].x += t1 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t1 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t1 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t1 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = head;
            I_do = 0.0;
            KSI = -1.0;
            return true;

        }
            
    }
    else if (mode == 4)
    {
        t2 = time;
        /*x = X + t2 * Vx;
        y = Y + t2 * Vy;
        z = Z + t2 * Vz;*/
        double l = sqrt(kvv(t2 * Vx, t2 * Vy, t2 * Vz));
        double alpha = polar_angle(Y, Z);
        double x = sqrt(kvv(Vx - vx, Vy - vy * cos(alpha), Vz - vy * sin(alpha)));
        uz = exp(-x * x) / sqrtpi + (x + 1.0 / (2.0 * x)) * erf(x);
        double sig = Kn * sqrt(kvv(Vx, Vy, Vz)) / (ro * uz * sigma(uz));
        double I = I_do + l / sig;
        if (I < KSI || ChEx == false)  // �� ��������� �����������
        {
            X = X + t2 * Vx;
            Y = Y + t2 * Vy;
            Z = Z + t2 * Vz;
            I_do = I;

            double alpha2 = polar_angle(Y - 0.5 * t2 * Vy, Z - 0.5 * t2 * Vz);
            
            mut.lock();
            nn1[head] += t2 * mu;
            //nn2[head].x += t2 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t2 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t2 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t2 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = (m-1) * N + n;

            if (m - 1 < 0)
            {
                cout << "Ne dolgen suda popadat" << endl;
                error = true;
                next = head;
                Vy = -Vy;
                Vz = -Vz;
                return true;
            }
            else
            {
                return true;
            }
        }
        else  // ����������� ����
        {
            double ksi = (KSI - I_do) * sig;
            t2 = ksi / sqrt(kvv(Vx, Vy, Vz));
            if (t2 > time)
            {
                cout << "ERROR   2872t42o4t28743234dsdwdsaeddaed" << endl;
            }
            X = X + t2 * Vx;
            Y = Y + t2 * Vy;
            Z = Z + t2 * Vz;

            double alpha2 = polar_angle(Y - 0.5 * t2 * Vy, Z - 0.5 * t2 * Vz);
            
            mut.lock();
            nn1[head] += t2 * mu;
            //nn2[head].x += t2 * Vx * mu;
            nn2[head].x += (Vx - vx) * mu;
            //nn2[head].y += t2 * (Vy * cos(alpha2) + Vz * sin(alpha2)) * mu;
            nn2[head].y += ((Vy * cos(alpha2) + Vz * sin(alpha2)) - vy) * mu;
            nn2[head].z += t2 * (-Vy * sin(alpha2) + Vz * cos(alpha2)) * mu;
            //nn3[head] += t2 * kvv(Vx, Vy, Vz) * mu;
            nn3[head] += 0.5 * (kvv(Vx, Vy, Vz) - (kv(vx) + kv(vy))) * mu;
            mut.unlock();

            next = head;
            I_do = 0.0;
            KSI = -1.0;
            return true;
        }
            
    }
    
    
    return false; 
}

double minplus(double x, double y)
{
    if (x < 0.00000000001 && y < 0.00000000001)
    {
        return -1.0;
    }
    else if(x < 0.00000000001)
    {
        return y;
    }
    else if (y < 0.00000000001)
    {
        return x;
    }
    else if (x < y)
    {
        return x;
    }
    else
    {
        return y;
    }
}

bool Peresechenie(const double& x0, const double& y0, const double& x, const double& y, const double& z, const double& Vx,//
    const double& Vy, const double& Vz, int& mode, double& t)
{
    mode = 0;
    double t1 = -1.0, t2 = -1.0, t3 = -1.0, t4 = -1.0, t5 = -1.0, t6 = -1.0;
    if (fabs(Vx) > 0.0000001) 
    {
        t1 = (x0 + dx / 2.0 - x) / Vx;
        t2 = (x0 - dx / 2.0 - x) / Vx;
    }
    if (kv(Vy) + kv(Vz) > 0.0000001)
    {
        peresich(y, z, Vy, Vz, y0 + dy / 2.0, t3, t4);
        peresich(y, z, Vy, Vz, y0 - dy / 2.0, t5, t6);
        t3 = minplus(t3, t4);
        t4 = minplus(t5, t6);
    }
    t = 100000000.0;
    if (t > t1 && t1 > 0.000000001)
    {
        t = t1;
        mode = 1;
    }
    if (t > t2 && t2 > 0.000000001)
    {
        t = t2;
        mode = 2;
    }
    if (t > t3 && t3 > 0.000000001)
    {
        t = t3;
        mode = 3;
    }
    if (t > t4 && t4 > 0.000000001)
    {
        t = t4;
        mode = 4;
    }

    if (fabs(Vx) > 0.0000001 && t1 < 0.000000001 && t2 < 0.000000001)
    {
        return false;
    }

    if(mode == 0)
    {
        return false;
    }
    return true;
}

void peresich(const double& y, const double& z, const double& Vy, const double& Vz, const double& R, double& t1, double& t2)
{
    double b = (2.0 * y * Vy + 2.0 * z * Vz);
    double a = (kv(Vy) + kv(Vz));
    double D = b * b - 4.0 * (kv(y) + kv(z) - kv(R)) * a;
    if (D < 0)
    {
        t1 = -1.0;
        t2 = -1.0;
        return;
    }
    D = sqrt(D);
    t1 = (-b + D) / (2.0 * a);
    t2 = (-b - D) / (2.0 * a);
}


void Velosity_initial(Sensor* s, double& Vx, double& Vy, double& Vz)
{
    double ksi1 = s->MakeRandom();
    double ksi2 = s->MakeRandom();
    double a = sqrt(-log(1.0 - ksi2));
    Vy = a * cos(2.0 * pi * ksi1);
    Vz = a * sin(2.0 * pi * ksi1);
    //cout << Vy << endl;
    double ksi3, ksi4, ksi5, ksi6;
    double z = 0;
    double p1 = fabs(Velosity_inf) * sqrtpi / (1.0 + fabs(Velosity_inf) * sqrtpi);

    do
    {
        ksi3 = s->MakeRandom();
        ksi4 = s->MakeRandom();
        ksi5 = s->MakeRandom();
        ksi6 = s->MakeRandom();

        if (p1 > ksi3)
        {
            z = cos(pi * ksi5) * sqrt(-log(ksi4));
        }
        else
        {
            if (ksi4 <= 0.5)
            {
                z = -sqrt(-log(2.0 * ksi4));
            }
            else
            {
                z = sqrt(-log(2.0 * (1.0 - ksi4)));
            }
        }
    } while (fabs(z + Velosity_inf) / (fabs(Velosity_inf) + fabs(z)) <= ksi6 || z > -Velosity_inf);

    Vx = z + Velosity_inf;
    return;
}

void Velosity_initial2(Sensor* s, double& Vx, double& Vy, double& Vz)
{
    double ksi1 = s->MakeRandom();
    double ksi2 = s->MakeRandom();
    double a = sqrt(-log(1.0 - ksi2));
    Vy = a * cos(2.0 * pi * ksi1);
    Vz = a * sin(2.0 * pi * ksi1);
    //cout << Vy << endl;
    double ksi3, ksi4, ksi5, ksi6;
    double z = 0;
    double p1 = 0.5 * fabs(Velosity_inf) * sqrtpi / (0.5 + 0.5 *  fabs(Velosity_inf) * sqrtpi);

    do
    {
        ksi3 = s->MakeRandom();
        ksi4 = s->MakeRandom();
        ksi5 = s->MakeRandom();
        ksi6 = s->MakeRandom();

        if (p1 > ksi3)
        {
            z = cos(pi * ksi5) * sqrt(-log(ksi4));
        }
        else
        {
            z = sqrt(-log(1.0 - ksi4));
            
        }
    } while (fabs(z + Velosity_inf) / (fabs(Velosity_inf) + fabs(z)) <= ksi6 || z < -Velosity_inf);

    Vx = z + Velosity_inf;
    if (Vx <= 0)
    {
        cout << "dfEEERR 32424442" << endl;
    }
    return;
}

void Change_Velosity(Sensor* s, const double& Ur, const double& Uthe, const double& Uphi, //
                        const double& Vr, const double& Vthe, const double& Vphi, double& X, double& Y, double& Z)
{
    //cout << ux << " " << uy << " " << uz << " " << Vx << " " << Vy << " " << Vz << endl;
    double x = sqrt(kvv(Vr - Ur, Vthe - Uthe, Vphi - Uphi));
    double p4 = 0.5 * sqrtpi * x / (1.0 + 0.5 * sqrtpi * x);
    /*watch(Vr);
    watch(Vthe);
    watch(Vphi);
    watch(Ur);
    watch(Uthe);
    watch(Uphi);
    watch(x);
    watch(p4);*/
    //exit(-1);
    //double p4 = sqrtpi * x / (1.0 + sqrtpi * x);
    //double p5 = 1.0 - p4;
    double ksi1, ksi2, ksi3, ksi4, ksi5, ksi6;
    double om1, om2, om3, lo;
    double y1, y2, y3;
    double v1, v2, v3, u1, u2, u3;
    double uu, yy, vv, D, ko;
    do
    {
        ksi1 = s->MakeRandom();
        ksi2 = s->MakeRandom();
        ksi3 = s->MakeRandom();
        ksi4 = s->MakeRandom();
        ksi5 = s->MakeRandom();
        ksi6 = s->MakeRandom();
        //cout << "sd " << endl;
        if (p4 < ksi1)
        {
            om1 = 1.0 - 2.0 * ksi4;
            om2 = sqrt(1.0 - kv(om1)) * cos(2.0 * pi * ksi5);
            om3 = sqrt(1.0 - kv(om1)) * sin(2.0 * pi * ksi5);
            // ����� ����������� ��������
            /*do 
            {
                om2 = 1.0 - 2.0 * s->MakeRandom();
                om3 = 1.0 - 2.0 * s->MakeRandom();
                D = kv(om2) + kv(om3);
            } while (D > 1);
            ko = sqrt((1.0 - kv(om1)) / D);
            om2 = om2 * ko;
            om3 = om3 * ko;*/

            lo = sqrt(-log(ksi2 * ksi3));
            y1 = lo * om1;
            y2 = lo * om2;
            y3 = lo * om3;
        }
        else
        {
            y1 = sqrt(-log(ksi2)) * cos(pi * ksi3);
            y2 = sqrt(-log(ksi4)) * cos(2.0 * pi * ksi5);
            y3 = sqrt(-log(ksi4)) * sin(2.0 * pi * ksi5);
        }
        v1 = y1 + Ur;
        v2 = y2 + Uthe;
        v3 = y3 + Uphi;
        u1 = Vr - v1;
        u2 = Vthe - v2;
        u3 = Vphi - v3;
        uu = sqrt(kvv(u1, u2, u3));
        yy = sqrt(kvv(y1, y2, y3));
    } while (  ( (uu * sigma(uu)) / (sigma(x) * (x + yy)) ) <= ksi6  );
    //cout << v2 << endl;
    X = v1;
    Y = v2;
    Z = v3;
}

void Belong_point(const double& x, const double& y, int& n, int& m)
{
    m = (int)(y / dy);
    n = (int)((x - x_min + dx/2.0) / dx);
    return;
}