#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <string>
#include "Header.h"
#include "sensor.h"

using namespace std;

//__device__ int sign(double& x);
__device__ double minmod(double x, double y);
__device__ double linear(double x1, double t1, double x2, double t2, double x3, double t3, double y);
__device__ void linear2(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
    double& A, double& B);
__global__ void add2(double2* s, double2* u, double2* s2, double2* u2, double* T, double* T_do, int method, int step);

__device__ double minmod(double x, double y)
{
    if (sign(x) + sign(y) == 0)
    {
        return 0.0;
    }
    else
    {
        return   ((sign(x) + sign(y)) / 2.0) * min(fabs(x), fabs(y));  ///minmod
        //return (2*x*y)/(x + y);   /// vanleer
    }
}

__device__ double linear(double x1, double t1, double x2, double t2, double x3, double t3, double y)
{
    double d = minmod((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    return  (d * (y - x2) + t2);
}

__device__ void linear2(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
    double& A, double& B)
{
    // ������� �������� - ����������� - �� ������� �� ����
    double d = minmod((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    A = (d * (y1 - x2) + t2);
    B = (d * (y2 - x2) + t2);
    //printf("%lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf \n", x1, t1, x2, t2, x3, t3, y1, y2, A, B);
    return;
}


__device__ double  my_min(double a, double b)
{
    if (a <= b)
    {
        return a;
    }
    else
    {
        return b;
    }
}

__device__ double  my_max(double a, double b)
{
    if (a >= b)
    {
        return a;
    }
    else
    {
        return b;
    }
}

__device__ void lev(const double& enI, const double& pI, const double& rI, const double& enII,//
    const double& pII, const double& rII, double& uuu, double& fee);
__device__ void devtwo(const double& enI, const double& pI, const double& rI, const double& enII, const double& pII, const double& rII, //
    const double& w, double& p);
__device__ void newton(const double& enI, const double& pI, const double& rI, const double& enII, const double& pII, const double& rII, //
    const double& w, double& p);
__device__ void perpendicular(double a1, double a2, double a3, double& b1, double& b2, double& b3, //
    double& c1, double& c2, double& c3, bool t);
__device__ double Godunov_Solver_Alexashov(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad);
__host__ bool areaa(double x, double y, double ro, double p, double u, double v);

__host__ bool areaa(double  x, double y, double ro, double p, double u, double v)
{
    if (ro <= 0.0)
    {
        return true;
    }
    double Max = sqrt((u * u + v * v) / (ggg * p / ro));
    double T = p / ro;
    if ((x < 36.8) && (y < 336))
    {
        return true;
    }
    if (( fabs(ro - 1.0) < 0.000001) && (fabs(Max - 3.0) < 0.000001))
    {
        return false;
    }
    if ((x > 240.3)||(y > 616.4) )
    {
        return false;
    }
    if ((x < -368) && ( T > 0.12))
    {
        return true;
    }
    if (M > 3.3)
    {
        return true;
    }
    if ((x > 1.0)&&(ro < 1.7))
    {
        return true;
    }
    else
    {
        return false;
    }
    return false;
}

__device__ double Godunov_Solver_Alexashov(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double w = 0.0;
    double al = n1;
    double be = n2;
    double ge = 0.0;
    double time = 0.0;

    double al2 = -n2;
    double be2 = n1;
    double ge2 = 0.0;
    double al3 = 0.0;
    double be3 = 0.0;
    double ge3 = 1.0;

    double enI = al * Lu.x + be * Lu.y;
    double teI2 = al2 * Lu.x + be2 * Lu.y;
    double teI3 = al3 * Lu.x + be3 * Lu.y;
    double enII = al * Ru.x + be * Ru.y;
    double teII2 = al2 * Ru.x + be2 * Ru.y;
    double teII3 = al3 * Ru.x + be3 * Ru.y;

    double pI = Ls.y;
    double pII = Rs.y;
    double rI = Ls.x;
    double rII = Rs.x;

    int ipiz = 0;
    if (pI > pII)   // ����� ������� �������
    {
        double eno2 = enII;;
        double teo22 = teII2;
        double teo23 = teII3;
        double p2 = pII;
        double r2 = rII;

        double eno1 = enI;
        double teo12 = teI2;
        double teo13 = teI3;
        double p1 = pI;
        double r1 = rI;

        enI = -eno2;
        teI2 = teo22;
        teI3 = teo23;
        pI = p2;
        rI = r2;

        enII = -eno1;
        teII2 = teo12;
        teII3 = teo13;
        pII = p1;
        rII = r1;
        w = -w;
        ipiz = 1;                                                                // ???? �� ����� ����� ������ ����?
    }

    double cI = 0.0;
    double cII = 0.0;
    if (rI != 0.0)
    {
        cI = __dsqrt_rn(ga * pI / rI);
    }
    if (rII != 0.0)
    {
        cII = __dsqrt_rn(ga * pII / rII);
    }

   /* printf("C2 !!!! = %lf =  kor  %lf \n", cII, ga * pII / rII);
    printf("%lf , %lf, %lf \n",ga,pII,rII);*/

    double a = __dsqrt_rn(rI * (g2 * pII + g1 * pI) / 2.0);
    double Uud = (pII - pI) / a;
    double Urz = -2.0 * cII / g1 * (1.0 - pow((pI / pII), gm));
    double Uvk = -2.0 * (cII + cI) / g1;
    double Udf = enI - enII;

    int il, ip;
    double p, r, te2, te3, en;

    if (Udf < Uvk)
    {
        il = -1;
        ip = -1;
    }
    else if ((Udf >= Uvk) && (Udf <= Urz))
    {
        p = pI * pow(((Udf - Uvk) / (Urz - Uvk)), (1.0 / gm));
        il = 0;
        ip = 0;
    }
    else if ((Udf > Urz) && (Udf <= Uud))
    {
        devtwo(enI, pI, rI, enII, pII, rII, w, p);
        il = 1;
        ip = 0;
    }
    else if (Udf > Uud)
    {
        newton(enI, pI, rI, enII, pII, rII, w, p);
        il = 1;
        ip = 1;
    }

    //*********TWO SHOCKS**********************************************
    if ((il == 1) && (ip == 1))
    {
       /* printf("TWO SHOCKS\n");*/
        double aI = __dsqrt_rn(rI * (g2 / 2.0 * p + g1 / 2.0 * pI));
        double aII = __dsqrt_rn(rII * (g2 / 2.0 * p + g1 / 2.0 * pII));

        double u = (aI * enI + aII * enII + pI - pII) / (aI + aII);
        double dI = enI - aI / rI;
        double dII = enII + aII / rII;


        double UU = max(fabs(dI), fabs(dII));
        if (UU > eps8)
        {
            time = krit * rad / UU;
        }
        else
        {
            time = krit * rad / eps8;
        }


        if (w <= dI)
        {
            en = enI;
            p = pI;
            r = rI;
            te2 = teI2;
            te3 = teI3;
        }
        else if ((w > dI) && (w <= u))
        {
            en = u;
            p = p;
            r = rI * aI / (aI - rI * (enI - u));
            te2 = teI2;
            te3 = teI3;
        }
        else if ((w > u) && (w < dII))
        {
            en = u;
            p = p;
            r = rII * aII / (aII + rII * (enII - u));
            te2 = teII2;
            te3 = teII3;
        }
        else if (w >= dII)
        {
            en = enII;
            p = pII;
            r = rII;
            te2 = teII2;
            te3 = teII3;
        }
    }


    //*********LEFT - SHOCK, RIGHT - EXPANSION FAN*******************
    if ((il == 1) && (ip == 0))
    {
        //printf("LEFT - SHOCK, RIGHT - EXPANSION FAN\n");
        double aI = __dsqrt_rn(rI * (g2 / 2.0 * p + g1 / 2.0 * pI));
        double aII;
        if (fabs(p - pII) < eps)
        {
            aII = rII * cII;
        }
        else
        {
            aII = gm * rII * cII * (1.0 - p / pII) / (1.0 - pow((p / pII), gm));
        }

        double u = (aI * enI + aII * enII + pI - pII) / (aI + aII);
        double dI = enI - aI / rI;
        double dII = enII + cII;
        double ddII = u + cII - g1 * (enII - u) / 2.0;

        double UU = max(fabs(dI), fabs(dII));
        UU = max(UU, fabs(ddII));
        if (UU > eps8)
        {
            time = krit * rad / UU;
        }
        else
        {
            time = krit * rad / eps8;
        }

        if (w <= dI)
        {
            en = enI;
            p = pI;
            r = rI;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w > dI) && (w <= u))
        {
            en = u;
            p = p;
            r = rI * aI / (aI - rI * (enI - u));
            te2 = teI2;
            te3 = teI3;
        }
        if ((w > u) && (w <= ddII))
        {
            double ce = cII - g1 / 2.0 * (enII - u);
            en = u;
            p = p;
            r = ga * p / ce / ce;
            te2 = teII2;
            te3 = teII3;
        }
        if ((w > ddII) && (w < dII))
        {
            double ce = -g1 / g2 * (enII - w) + 2.0 / g2 * cII;
            en = w - ce;
            p = pII * pow((ce / cII), (1.0 / gm));
            r = ga * p / ce / ce;
            te2 = teII2;
            te3 = teII3;
        }
        if (w >= dII)
        {
            en = enII;
            p = pII;
            r = rII;
            te2 = teII2;
            te3 = teII3;
        }
    }
    //*********TWO EXPANSION FANS**************************************
    if ((il == 0) && (ip == 0))
    {
        //printf("TWO EXPANSION FANS\n");
        double aI;
        //printf("p = %lf\n", p);
        if (fabs(p - pI) < eps)
        {
            aI = rI * cI;
        }
        else
        {
            aI = gm * rI * cI * (1.0 - p / pI) / (1.0 - pow((p / pI), gm));
        }
        //printf("aI = %lf\n", aI);

        double aII;
        if (fabs(p - pII) < eps)
        {
            aII = rII * cII;
        }
        else
        {
            aII = gm * rII * cII * (1.0 - p / pII) / (1.0 - pow((p / pII), gm));
        }

        //printf("aII = %lf\n", aI);

        double u = (aI * enI + aII * enII + pI - pII) / (aI + aII);
        double dI = enI - cI;
        double ddI = u - cI - g1 * (enI - u) / 2.0;
        double dII = enII + cII;
        double ddII = u + cII - g1 * (enII - u) / 2.0;
        /*printf("enII = %lf\n", enII);
        printf("cII = %lf\n", cII);
        printf("u = %lf\n", u);
        printf("dI = %lf\n", dI);
        printf("dII = %lf\n", dII);
        printf("ddI = %lf\n", ddI);
        printf("ddII = %lf\n", ddII);*/

        double UU = max(fabs(dI), fabs(dII));
        UU = max(UU, fabs(ddII));
        UU = max(UU, fabs(ddI));
        if (UU > eps8)
        {
            time = krit * rad / UU;
        }
        else
        {
            time = krit * rad / eps8;
        }


        if (w <= dI)
        {
            //printf("1\n");
            en = enI;
            p = pI;
            r = rI;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w > dI) && (w < ddI))
        {
            //printf("2\n");
            double ce = g1 / g2 * (enI - w) + 2.0 / g2 * cI;
            en = w + ce;
            p = pI * pow((ce / cI), (1.0 / gm));
            r = ga * p / ce / ce;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w >= ddI) && (w <= u))
        {
            //printf("3\n");
            double ce = cI + g1 / 2.0 * (enI - u);
            en = u;
            p = p;
            r = ga * p / ce / ce;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w > u) && (w <= ddII))
        {
            //printf("4\n");
            double ce = cII - g1 / 2.0 * (enII - u);
            en = u;
            p = p;
            r = ga * p / ce / ce;
            te2 = teII2;
            te3 = teII3;
        }
        if ((w > ddII) && (w < dII))
        {
            //printf("5\n");
            double ce = -g1 / g2 * (enII - w) + 2.0 / g2 * cII;
            en = w - ce;
            p = pII * pow((ce / cII), (1.0 / gm));
            r = ga * p / ce / ce;
            te2 = teII2;
            te3 = teII3;
        }
        if (w >= dII)
        {
            //printf("6\n");
            en = enII;
            p = pII;
            r = rII;
            te2 = teII2;
            te3 = teII3;
        }
    }

    //*********VAKUUM ************************************************
    if ((il == -1) && (ip == -1))
    {
        //printf("VAKUUM\n");
        double dI = enI - cI;
        double ddI = enI + 2.0 / gg1 * cI;
        double dII = enII + cII;
        double ddII = enII - 2.0 / gg1 * cII;


        double UU = max(fabs(dI), fabs(dII));
        UU = max(UU, fabs(ddII));
        UU = max(UU, fabs(ddI));
        if (UU > eps8)
        {
            time = krit * rad / UU;
        }
        else
        {
            time = krit * rad / eps8;
        }


        if (w <= dI)
        {
            en = enI;
            p = pI;
            r = rI;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w > dI) && (w < ddI))
        {
            double ce = gg1 / gg2 * (enI - w) + 2.0 / gg2 * cI;
            en = w + ce;
            p = pI * pow((ce / cI), (1.0 / gm));
            r = gga * p / ce / ce;
            te2 = teI2;
            te3 = teI3;
        }
        if ((w >= ddI) && (w <= ddII))
        {
            en = w;
            p = 0.0;
            r = 0.0;
            te2 = 0.0;
            te3 = 0.0;
        }
        if ((w > ddII) && (w < dII))
        {
            double ce = -gg1 / gg2 * (enII - w) + 2.0 / gg2 * cII;
            en = w - ce;
            p = pII * pow((ce / cII), (1.0 / gm));
            r = gga * p / ce / ce;
            te2 = teII2;
            te3 = teII3;
        }
        if (w >= dII)
        {
            en = enII;
            p = pII;
            r = rII;
            te2 = teII2;
            te3 = teII3;
        }
    }


    if (ipiz == 1)
    {
        en = -en;
        w = -w;
    }

    double uo = al * en + al2 * te2 + al3 * te3;
    double vo = be * en + be2 * te2 + be3 * te3;
    double wo = ge * en + ge2 * te2 + ge3 * te3;


    double eo = p / g1 + 0.5 * r * (uo * uo + vo * vo + wo * wo);
    en = al * uo + be * vo + ge * wo;

    Ps.x = (r * (en - w));
    Pu.x = (r * (en - w) * uo + al * p);
    Pu.y = (r * (en - w) * vo + be * p);
    //qqq[3] = (r * (en - w) * wo + ge * p);
    Ps.y = ((en - w) * eo + en * p);


    return time;

}

__device__ void perpendicular(double a1, double a2, double a3, double& b1, double& b2, double& b3, //
    double& c1, double& c2, double& c3, bool t)
{
    if (t == false)
    {
        double A = a1 * a1 + a2 * a2;
        if (A > 0.01 * (A + a3 * a3))
        {
            double B = sqrt(A);
            b1 = -a2 / B;
            b2 = a1 / B;
            b3 = 0.0;
            double C = sqrt(A * (A + a3 * a3));
            c1 = -a1 * a3 / C;
            c2 = -a2 * a3 / C;
            c3 = A / C;
            return;
        }
        A = a1 * a1 + a3 * a3;
        if (A > 0.01 * (A + a2 * a2))
        {
            double B = sqrt(A);
            b1 = -a3 / B;
            b2 = 0.0;
            b3 = a1 / B;
            double C = sqrt(A * (A + a2 * a2));
            c1 = a1 * a2 / C;
            c2 = -A / C;
            c3 = a2 * a3 / C;
            return;
        }
    }
    else
    {
        double A = a1 * a1 + a2 * a2;
        if (A > 0.01)
        {
            double B = sqrt(A);
            b1 = -a2 / B;
            b2 = a1 / B;
            b3 = 0.0;;
            c1 = -a1 * a3 / B;
            c2 = -a2 * a3 / B;
            c3 = A / B;
            return;
        }
        A = a1 * a1 + a3 * a3;
        if (A > 0.01)
        {
            double B = sqrt(A);
            b1 = -a3 / B;
            b2 = 0.0;
            b3 = a1 / B;

            c1 = a1 * a2 / B;
            c2 = -A / B;
            c3 = a2 * a3 / B;
            return;
        }
    }

}

__device__ void newton(const double& enI, const double& pI, const double& rI, const double& enII, const double& pII, const double& rII, //
    const double& w, double& p)
{
    double fI, fIs, fII, fIIs;
    double cI = __dsqrt_rn(ga * pI / rI);
    double cII = __dsqrt_rn(ga * pII / rII);
    double pn = pI * rII * cII + pII * rI * cI + (enI - enII) * rI * cI * rII * cII;
    pn = pn / (rI * cI + rII * cII);

    double pee = pn;

    int kiter = 0;
a1:
    p = pn;
    if (p <= 0.0)
    {
        printf("84645361\n");
    }

    kiter = kiter + 1;

    fI = (p - pI) / (rI * cI * __dsqrt_rn(gp * p / pI + gm));
    fIs = (ga + 1.0) * p / pI + (3.0 * ga - 1.0);
    fIs = fIs / (4.0 * ga * rI * cI * pow((gp * p / pI + gm), (3.0 / 2.0)));

    fII = (p - pII) / (rII * cII * __dsqrt_rn(gp * p / pII + gm));
    fIIs = (ga + 1.0) * p / pII + (3.0 * ga - 1.0);
    fIIs = fIIs / (4.0 * ga * rII * cII * pow((gp * p / pII + gm), (3.0 / 2.0)));


    if (kiter == 1100)
    {
        printf("0137592\n");
    }

    pn = p - (fI + fII - (enI - enII)) / (fIs + fIIs);

    if (fabs(pn / pee - p / pee) >= eps)
    {
        goto a1;
    }

    p = pn;

    return;
}

__device__ void devtwo(const double& enI, const double& pI, const double& rI, const double& enII, const double& pII, const double& rII, //
    const double& w, double& p)
{
    const double epsil = 10e-10;
    double kl, kp, kc, ksi, ksir, um, ksit;
    int kpizd;

    kl = pI;
    kp = pII;


    lev(enI, pI, rI, enII, pII, rII, kl, ksi);
    lev(enI, pI, rI, enII, pII, rII, kp, ksir);

    if (fabs(ksi) <= epsil)
    {
        um = kl;
        goto a1;
    }

    if (fabs(ksir) <= epsil)
    {
        um = kp;
        goto a1;
    }

    kpizd = 0;

a2:
    kpizd = kpizd + 1;

    if (kpizd == 1100)
    {
        printf("121421414\n");
        printf("%lf, %lf,%lf,%lf,%lf,%lf,\n", enI, pI, rI, enII, pII, rII);
    }


    kc = (kl + kp) / 2.0;

    lev(enI, pI, rI, enII, pII, rII, kc, ksit);

    if (fabs(ksit) <= epsil)
    {
        goto a3;
    }

    if ((ksi * ksit) <= 0.0)
    {
        kp = kc;
        ksir = ksit;
    }
    else
    {
        kl = kc;
        ksi = ksit;
    }

    goto a2;

a3:
    um = kc;
a1:

    p = um;

    return;
}

__device__ void lev(const double& enI, const double& pI, const double& rI, const double& enII,//
    const double& pII, const double& rII, double& uuu, double& fee)
{
    double cI = __dsqrt_rn(ga * pI / rI);
    double cII = __dsqrt_rn(ga * pII / rII);

    double fI = (uuu - pI) / (rI * cI * __dsqrt_rn(gp * uuu / pI + gm));

    double fII = 2.0 / g1 * cII * (pow((uuu / pII), gm) - 1.0);

    double f1 = fI + fII;
    double f2 = enI - enII;
    fee = f1 - f2;
    return;
}

__device__ double HLLC_Aleksashov(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double n[3];
    n[0] = n1;
    n[1] = n2;
    n[2] = 0.0;
    //int id_bn = 1;
    //int n_state = 1;
    double FR[8], FL[8];
    double UL[8], UZ[8], UR[8];
    double UZL[8], UZR[8];

    double vL[3], vR[3], bL[3], bR[3];
    double vzL[3], vzR[3], bzL[3], bzR[3];
    double qv[3];
    double aco[3][3];

    double wv = 0.0;
    double r1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double w1 = 0.0;
    double p1 = Ls.y;
    double bx1 = 0.0;
    double by1 = 0.0;
    double bz1 = 0.0;


    double r2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double w2 = 0.0;
    double p2 = Rs.y;
    double bx2 = 0.0;
    double by2 = 0.0;
    double bz2 = 0.0;

    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;
    double abx = (bx2 + bx1) / 2.0;
    double aby = (by2 + by1) / 2.0;
    double abz = (bz2 + bz1) / 2.0;


    double bk = abx * n[0] + aby * n[1] + abz * n[2];
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double d = b2 - kv(bk);
    aco[0][0] = n[0];
    aco[1][0] = n[1];
    aco[2][0] = n[2];
    if (d > eps)
    {
        d = __dsqrt_rn(d);
        aco[0][1] = (abx - bk * n[0]) / d;
        aco[1][1] = (aby - bk * n[1]) / d;
        aco[2][1] = (abz - bk * n[2]) / d;
        aco[0][2] = (aby * n[2] - abz * n[1]) / d;
        aco[1][2] = (abz * n[0] - abx * n[2]) / d;
        aco[2][2] = (abx * n[1] - aby * n[0]) / d;
    }
    else
    {
        double aix, aiy, aiz;
        if ((fabs(n[0]) < fabs(n[1])) && (fabs(n[0]) < fabs(n[2])))
        {
            aix = 1.0;
            aiy = 0.0;
            aiz = 0.0;
        }
        else if (fabs(n[1]) < fabs(n[2]))
        {
            aix = 0.0;
            aiy = 1.0;
            aiz = 0.0;
        }
        else
        {
            aix = 0.0;
            aiy = 0.0;
            aiz = 1.0;
        }

        double aik = aix * n[0] + aiy * n[1] + aiz * n[2];
        d = __dsqrt_rn(1.0 - kv(aik));
        aco[0][1] = (aix - aik * n[0]) / d;
        aco[1][1] = (aiy - aik * n[1]) / d;
        aco[2][1] = (aiz - aik * n[2]) / d;
        aco[0][2] = (aiy * n[2] - aiz * n[1]) / d;
        aco[1][2] = (aiz * n[0] - aix * n[2]) / d;
        aco[2][2] = (aix * n[1] - aiy * n[0]) / d;
    }

    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }

    double aaL = bL[0] / __dsqrt_rn(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = __dsqrt_rn(ga * p1 / r1);
    double qp = __dsqrt_rn(b21 + cL * (cL + 2.0 * aaL));
    double qm = __dsqrt_rn(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / 2.0;
    double ptL = p1 + b2L / 2.0;

    double aaR = bR[0] / __dsqrt_rn(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = __dsqrt_rn(ga * p2 / r2);
    qp = __dsqrt_rn(b22 + cR * (cR + 2.0 * aaR));
    qm = __dsqrt_rn(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / 2.0;
    double ptR = p2 + b2R / 2.0;

    double aC = (aaL + aaR) / 2.0;
    double b2o = (b22 + b21) / 2.0;
    double cC = __dsqrt_rn(ga * ap / ro);
    qp = __dsqrt_rn(b2o + cC * (cC + 2.0 * aC));
    qm = __dsqrt_rn(b2o + cC * (cC - 2.0 * aC));
    double cfC = (qp + qm) / 2.0;
    double vC1 = (vL[0] + vR[0]) / 2.0;

    double SL = min((vL[0] - cfL), (vR[0] - cfR));
    double SR = max((vL[0] + cfL), (vR[0] + cfR));

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);

    if (SR <= SL)
    {
        printf("231\n");
    }

    double SM00 = SM;
    double SR00 = SR;
    double SL00 = SL;
    double SM01, SR01, SL01;
    if ((SM00 >= SR00) || (SM00 <= SL00))
    {
        SL = min((vL[0] - cfL), (vR[0] - cfR));
        SR = max((vL[0] + cfL), (vR[0] + cfR));
        suR = SR - vR[0];
        suL = SL - vL[0];
        SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);
        SM01 = SM;
        SR01 = SR;
        SL01 = SL;
        if ((SM01 >= SR01) || (SM01 <= SL01))
        {
            printf("251\n");
        }
    }


    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / 2.0;
    double e2 = p2 / g1 + r2 * upt2 + b2R / 2.0;

    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;


    for (int ik = 0; ik < 3; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UL[ik + 5] = bL[ik];
        UR[ik + 1] = r2 * vR[ik];
        UR[ik + 5] = bR[ik];
    }

    for (int ik = 0; ik < 8; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }

    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = ptR + r2 * suR * (SM - vR[0]);
    double ptzL = ptL + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;
    bzR[0] = UZ[5];
    bzL[0] = UZ[5];

    vzR[1] = UZ[2] / UZ[0];
    vzR[2] = UZ[3] / UZ[0];
    vzL[1] = vzR[1];
    vzL[2] = vzR[2];

    vzR[1] = vR[1] + UZ[5] * (bR[1] - UZ[6]) / suR / r2;
    vzR[2] = vR[2] + UZ[5] * (bR[2] - UZ[7]) / suR / r2;
    vzL[1] = vL[1] + UZ[5] * (bL[1] - UZ[6]) / suL / r1;
    vzL[2] = vL[2] + UZ[5] * (bL[2] - UZ[7]) / suL / r1;

    bzR[1] = UZ[6];
    bzR[2] = UZ[7];
    bzL[1] = bzR[1];
    bzL[2] = bzR[2];

    double sbvz = (UZ[5] * UZ[1] + UZ[6] * UZ[2] + UZ[7] * UZ[3]) / UZ[0];

    double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + UZ[5] * (sbv2 - sbvz)) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + UZ[5] * (sbv1 - sbvz)) / (SL - SM);

    if (fabs(UZ[5]) < eps)
    {
        vzR[1] = vR[1];
        vzR[2] = vR[2];
        vzL[1] = vL[1];
        vzL[2] = vL[2];
        bzR[1] = bR[1] * suRm;
        bzR[2] = bR[2] * suRm;
        bzL[1] = bL[1] * suLm;
        bzL[2] = bL[2] * suLm;
    }
    UZL[0] = rzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[4] = ezR;

    for (int ik = 0; ik < 3; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZL[ik + 5] = bzL[ik];
        UZR[ik + 1] = vzR[ik] * rzR;
        UZR[ik + 5] = bzR[ik];
    }

    if (SL > wv)
    {
        Ps.x = FL[0] - wv * UL[0];
        Ps.y = FL[4] - wv * UL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] - wv * UL[ik];
        }
    }
    else if ( (SL <= wv) && (SM >= wv) )
    {
        Ps.x = FL[0] + SL * (rzL - r1) - wv * UZL[0];
        Ps.y = FL[4] + SL * (ezL - e1) - wv * UZL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
        }
    }
    else if ((SM <= wv)&&(SR >= wv))
    {
        Ps.x = FR[0] + SR * (rzR - r2) - wv * UZR[0];
        Ps.y = FR[4] + SR * (ezR - e2) - wv * UZR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
        }
    }
    else if (SR < wv)
    {
        Ps.x = FR[0] - wv * UR[0];
        Ps.y = FR[4] - wv * UR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + - wv * UR[ik];
        }
    }
    else
    {
        printf("DDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDD\n");
    }


    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1] + aco[0][2] * qv[2];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1] + aco[1][2] * qv[2];

    return time;
}

__device__ double HLLC_Aleksashov2(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double r1 = Ls.x;
    double p1 = Ls.y;
    double u1 = Lu.x;
    double v1 = Lu.y;

    double r2 = Rs.x;
    double p2 = Rs.y;
    double u2 = Ru.x;
    double v2 = Ru.y;



    // c------ - n_state = 2 - two - state(3 speed) HLLC(Contact Discontinuity)


    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;

    double aco[2][2];
    aco[0][0] = n1;
    aco[1][0] = n2;
    aco[0][1] = -n2;
    aco[1][1] = n1;

    //aco(1, 1) = al
    //aco(2, 1) = be
    //aco(3, 1) = ge

    double vL[2];
    double vR[2];

    vL[0] = aco[0][0] * u1 + aco[1][0] * v1;
    vL[1] = aco[0][1] * u1 + aco[1][1] * v1;
    vR[0] = aco[0][0] * u2 + aco[1][0] * v2;
    vR[1] = aco[0][1] * u2 + aco[1][1] * v2;

    /*if ((r1 <= eps) || (r2 <= eps) || (p1 <= 0) || (p2 <= 0) )
    {
        printf("EREREREEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEE\n");
    }*/

    double cL = __dsqrt_rn(ga * p1 / r1);
    double cR = __dsqrt_rn(ga * p2 / r2);
    double cC = __dsqrt_rn(ga * ap / ro);

    double SL, SR;

    //SL = min((vL[0] - cL), (vC1 - cC));
    //SR = max((vR[0] + cR), (vC1 + cC));

    SL = min((vL[0] - cL), (vR[0] - cR));
    SR = max((vL[0] + cL), (vR[0] + cR));

    double t = 10000000;
    t = min(t, krit * rad / max(fabs(SL), fabs(SR)));

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = 0.0;
    if (fabs(suR * r2 - suL * r1) > 0)
    {
        SM = (suR * r2 * vR[0] - p2 + p1 - suL * r1 * vL[1]) / (suR * r2 - suL * r1);
    }

    if (SR < SL)
    {
        printf("12102022020,    ERROR in HLCC_Alexashov  \n");
    }

    double upt1 = (u1 * u1 + v1 * v1) / 2.0;
    double upt2 = (u2 * u2 + v2 * v2) / 2.0;
    double e1 = p1 / g1 + r1 * upt1;
    double e2 = p2 / g1 + r2 * upt2;
    double FL[4];
    double FR[4];
    double UL[4];
    double UR[4];

    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + p1;
    FL[2] = r1 * vL[0] * vL[1];
    FL[3] = (e1 + p1) * vL[0];

    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + p2;
    FR[2] = r2 * vR[0] * vR[1];
    FR[3] = (e2 + p2) * vR[0];

    UL[0] = r1;
    UL[3] = e1;
    UR[0] = r2;
    UR[3] = e2;

    UL[1] = r1 * vL[0];
    UL[2] = r1 * vL[1];
    UR[1] = r2 * vR[0];
    UR[2] = r2 * vR[1];


    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;

    double ptzR = p2 + r2 * suR * (SM - vR[0]);
    double ptzL = p1 + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;
    double vzR[2];
    double vzL[2];

    vzR[0] = SM;
    vzL[0] = SM;
    vzR[1] = vR[1];
    vzL[1] = vL[1];

    double ezR = e2 * suRm + (ptz * SM - p2 * vR[0]) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - p1 * vL[0]) / (SL - SM);

    double UZL[4];
    double UZR[4];

    UZL[0] = rzL;
    UZL[3] = ezL;
    UZR[0] = rzR;
    UZR[3] = ezR;

    for (int i = 1; i < 3; i++)
    {
        UZL[i] = vzL[i - 1] * rzL;
        UZR[i] = vzR[i - 1] * rzR;
    }

    double qv[2];

    if (SL > 0.0)
    {
        Ps.x = FL[0];
        Ps.y = FL[3];
        qv[0] = FL[1];
        qv[1] = FL[2];
    }
    else if ((SL <= 0.0) && (SM >= 0.0))
    {
        Ps.x = FL[0] + SL * (rzL - r1);
        Ps.y = FL[3] + SL * (ezL - e1);
        qv[0] = FL[1] + SL * (UZL[1] - UL[1]);
        qv[1] = FL[2] + SL * (UZL[2] - UL[2]);
    }
    else if ((SM <= 0.0) && (SR >= 0.0))
    {
        Ps.x = FR[0] + SR * (rzR - r2);
        Ps.y = FR[3] + SR * (ezR - e2);
        qv[0] = FR[1] + SR * (UZR[1] - UR[1]);
        qv[1] = FR[2] + SR * (UZR[2] - UR[2]);
    }
    else if (SR < 0.0)
    {
        Ps.x = FR[0];
        Ps.y = FR[3];
        qv[0] = FR[1];
        qv[1] = FR[2];
    }
    else
    {
        printf("21702022020,    ERROR in HLCC_Alexashov  \n");
        printf(" SL = %lf, SM = %lf, SR = %lf\n", SL, SM, SR);
        printf(" r1 = %lf, p1 = %lf, u1 = %lf, v1 = %lf\n", r1, p1, u1, v1);
        printf(" r2 = %lf, p2 = %lf, u2 = %lf, v2 = %lf\n", r2, p2, u2, v2);
        printf(" vl[0] = %lf, cL = %lf, vR[0] = %lf, cR = %lf\n", vL[0], cL, vR[0], cR);
        /*SL = min((vL[0] - cL), (vR[0] - cR));
        SR = max((vL[0] + cL), (vR[0] + cR));*/
    }

    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1];

    return t;
}

__device__ double HLLC_Korolkov(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double ro_L = Ls.x;
    double p_L = Ls.y;
    double v1_L = Lu.x;
    double v2_L = Lu.y;

    double ro_R = Rs.x;
    double p_R = Rs.y;
    double v1_R = Ru.x;
    double v2_R = Ru.y;

    double e_L, e_R;
    double Vkv_L, Vkv_R;
    double c_L, c_R;

    Vkv_L = v1_L * v1_L + v2_L * v2_L;
    Vkv_R = v1_R * v1_R + v2_R * v2_R;

    c_L = __dsqrt_rn(ggg * p_L / ro_L);
    c_R = __dsqrt_rn(ggg * p_R / ro_R);
    e_L = p_L / (ggg - 1.0) + ro_L * Vkv_L / 2.0;  /// ������ ������� �����
    e_R = p_R / (ggg - 1.0) + ro_R * Vkv_R / 2.0;  /// ������ ������� ������

    double Vn_L = v1_L * n1 + v2_L * n2;
    double Vn_R = v1_R * n1 + v2_R * n2;

    double D_L = min(Vn_L, Vn_R) - max(c_L, c_R);
    double D_R = max(Vn_L, Vn_R) + max(c_L, c_R);
    /*double D_L = min(Vn_L - c_L, Vn_R - c_R);
    double D_R = max(Vn_L + c_L, Vn_R + c_R);*/
    double t = 10000000;
    t = min(t, krit * rad / max(fabs(D_L), fabs(D_R)));

    double fx1 = ro_L * v1_L;
    double fx2 = ro_L * v1_L * v1_L + p_L;
    double fx3 = ro_L * v1_L * v2_L;
    double fx5 = (e_L + p_L) * v1_L;

    double fy1 = ro_L * v2_L;
    double fy2 = ro_L * v1_L * v2_L;
    double fy3 = ro_L * v2_L * v2_L + p_L;
    double fy5 = (e_L + p_L) * v2_L;

    double fl_1 = fx1 * n1 + fy1 * n2;
    double fl_2 = fx2 * n1 + fy2 * n2;
    double fl_3 = fx3 * n1 + fy3 * n2;
    double fl_5 = fx5 * n1 + fy5 * n2;

    if (D_L > Omega)
    {
        Ps.x = fl_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
        Pu.x = fl_2;
        Pu.y = fl_3;
        Ps.y = fl_5;
        return t;
    }

    double hx1 = ro_R * v1_R;
    double hx2 = ro_R * v1_R * v1_R + p_R;
    double hx3 = ro_R * v1_R * v2_R;
    double hx5 = (e_R + p_R) * v1_R;

    double hy1 = ro_R * v2_R;
    double hy2 = ro_R * v1_R * v2_R;
    double hy3 = ro_R * v2_R * v2_R + p_R;
    double hy5 = (e_R + p_R) * v2_R;

    double fr_1 = hx1 * n1 + hy1 * n2;
    double fr_2 = hx2 * n1 + hy2 * n2;
    double fr_3 = hx3 * n1 + hy3 * n2;
    double fr_5 = hx5 * n1 + hy5 * n2;

    if (D_R < Omega)
    {
        Ps.x = fr_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
        Pu.x = fr_2;
        Pu.y = fr_3;
        Ps.y = fr_5;
        return t;
    }

    double u_L = Vn_L;
    double u_R = Vn_R;

    double D_C = ((D_R - u_R) * ro_R * u_R - (D_L - u_L) * ro_L * u_L - p_R + p_L) / ((D_R - u_R) * ro_R - (D_L - u_L) * ro_L);

    double roro_L = ro_L * ((D_L - u_L) / (D_L - D_C));
    double roro_R = ro_R * ((D_R - u_R) / (D_R - D_C));

    /// ������� �������� � ����������� ������� (��� ���������� ����� � ������)
    double P_T = (p_L * ro_R * (u_R - D_R) - p_R * ro_L * (u_L - D_L) - ro_L * ro_R * (u_L - D_L) * (u_R - D_R) * (u_R - u_L)) / (ro_R * (u_R - D_R) - ro_L * (u_L - D_L));

    if (D_L <= Omega && D_C >= Omega)  /// ������ �� ������ ������� (�����)
    {
        double Vx = v1_L + (D_C - Vn_L) * n1;
        double Vy = v2_L + (D_C - Vn_L) * n2;

        double ee_L = P_T / (ggg - 1.0) + roro_L * (Vx * Vx + Vy * Vy) / 2.0;
        //double ee_L = e_L - ((P_T - p_L)/2.0)*(1/roro_L - 1/ro_L);
        /*double ee_L = ((D_L - u_L) * e_L - p_L * u_L + P_T * D_C) / (D_L - D_C);*/

        double dq1 = roro_L - ro_L;
        double dq2 = roro_L * Vx - ro_L * v1_L;
        double dq3 = roro_L * Vy - ro_L * v2_L;
        double dq5 = ee_L - e_L;

        Ps.x = D_L * dq1 + fl_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
        Pu.x = D_L * dq2 + fl_2;
        Pu.y = D_L * dq3 + fl_3;
        Ps.y = D_L * dq5 + fl_5;
        return t;
    }
    else if (D_R >= Omega && D_C <= Omega)  /// ������ �� ������ ������� (������)
    {
        double Vx = v1_R + (D_C - Vn_R) * n1;
        double Vy = v2_R + (D_C - Vn_R) * n2;

        double ee_R = P_T / (ggg - 1.0) + roro_R * (Vx * Vx + Vy * Vy) / 2.0;
        /*double ee_R = ((D_R - u_R) * e_R - p_R * u_R + P_T * D_C) / (D_R - D_C);*/

        double dq1 = roro_R - ro_R;
        double dq2 = roro_R * Vx - ro_R * v1_R;
        double dq3 = roro_R * Vy - ro_R * v2_R;
        double dq5 = ee_R - e_R;

        Ps.x = D_R * dq1 + fr_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
        Pu.x = D_R * dq2 + fr_2;
        Pu.y = D_R * dq3 + fr_3;
        Ps.y = D_R * dq5 + fr_5;
        return t;
    }
    return t;
}

__device__ double HLL(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double ro_L = Ls.x;
    double p_L = Ls.y;
    double v1_L = Lu.x;
    double v2_L = Lu.y;

    double ro_R = Rs.x;
    double p_R = Rs.y;
    double v1_R = Ru.x;
    double v2_R = Ru.y;

    double e_L, e_R;
    double Vkv_L, Vkv_R;
    double c_L, c_R;

    Vkv_L = v1_L * v1_L + v2_L * v2_L;
    Vkv_R = v1_R * v1_R + v2_R * v2_R;
    if (ro_L <= 0)
    {
        c_L = 0.0;
    }
    else
    {
        c_L = sqrt(ggg * p_L / ro_L);
    }

    if (ro_R <= 0)
    {
        c_R = 0.0;
    }
    else
    {
        c_R = sqrt(ggg * p_R / ro_R);
    }
    e_L = p_L / (ggg - 1.0) + ro_L * Vkv_L / 2.0;  /// ������ ������� �����
    e_R = p_R / (ggg - 1.0) + ro_R * Vkv_R / 2.0;  /// ������ ������� ������

    double Vn_L = v1_L * n1 + v2_L * n2;
    double Vn_R = v1_R * n1 + v2_R * n2;
    double D_L = my_min(Vn_L, Vn_R) - my_max(c_L, c_R);
    double D_R = my_max(Vn_L, Vn_R) + my_max(c_L, c_R);
    double t = 10000000;
    t = my_min(t, krit * rad / my_max(fabs(D_L), fabs(D_R)));

    double fx1 = ro_L * v1_L;
    double fx2 = ro_L * v1_L * v1_L + p_L;
    double fx3 = ro_L * v1_L * v2_L;
    double fx5 = (e_L + p_L) * v1_L;

    double fy1 = ro_L * v2_L;
    double fy2 = ro_L * v1_L * v2_L;
    double fy3 = ro_L * v2_L * v2_L + p_L;
    double fy5 = (e_L + p_L) * v2_L;

    double fl_1 = fx1 * n1 + fy1 * n2;
    double fl_2 = fx2 * n1 + fy2 * n2;
    double fl_3 = fx3 * n1 + fy3 * n2;
    double fl_5 = fx5 * n1 + fy5 * n2;

    /*double U_L1 = ro_L;
    double U_L2 = ro_L * v1_L;
    double U_L3 = ro_L * v2_L;
    double U_L5 = e_L;*/

    if (D_L > Omega)
    {
        Ps.x = fl_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
        Pu.x = fl_2;
        Pu.y = fl_3;
        Ps.y = fl_5;
        return t;
    }
    else
    {
        double hx1 = ro_R * v1_R;
        double hx2 = ro_R * v1_R * v1_R + p_R;
        double hx3 = ro_R * v1_R * v2_R;
        double hx5 = (e_R + p_R) * v1_R;

        double hy1 = ro_R * v2_R;
        double hy2 = ro_R * v1_R * v2_R;
        double hy3 = ro_R * v2_R * v2_R + p_R;
        double hy5 = (e_R + p_R) * v2_R;

        double fr_1 = hx1 * n1 + hy1 * n2;
        double fr_2 = hx2 * n1 + hy2 * n2;
        double fr_3 = hx3 * n1 + hy3 * n2;
        double fr_5 = hx5 * n1 + hy5 * n2;

        /*double U_R1 = ro_R;
        double U_R2 = ro_R * v1_R;
        double U_R3 = ro_R * v2_R;
        double U_R5 = e_R;*/

        if (D_R < Omega)
        {
            Ps.x = fr_1; /// ����� ����� ��������� �� ������� ����� � ��� �� �������
            Pu.x = fr_2;
            Pu.y = fr_3;
            Ps.y = fr_5;
            return t;
        }
        else
        {
            double dq1 = ro_R - ro_L;
            double dq2 = ro_R * v1_R - ro_L * v1_L;
            double dq3 = ro_R * v2_R - ro_L * v2_L;
            double dq5 = e_R - e_L;

            //double U1 = (D_R * U_R1 - D_L * U_L1 - fr_1 + fl_1) / (D_R - D_L);
            //double U2 = (D_R * U_R2 - D_L * U_L2 - fr_2 + fl_2) / (D_R - D_L);
            //double U3 = (D_R * U_R3 - D_L * U_L3 - fr_3 + fl_3) / (D_R - D_L);
            //double U5 = (D_R * U_R5 - D_L * U_L5 - fr_5 + fl_5) / (D_R - D_L);


            Ps.x = (D_R * fl_1 - D_L * fr_1 + D_L * D_R * dq1) / (D_R - D_L); /// ����� ����� ��������� �� ������� ����� � ��� �� �������
            Pu.x = (D_R * fl_2 - D_L * fr_2 + D_L * D_R * dq2) / (D_R - D_L);
            Pu.y = (D_R * fl_3 - D_L * fr_3 + D_L * D_R * dq3) / (D_R - D_L);
            Ps.y = (D_R * fl_5 - D_L * fr_5 + D_L * D_R * dq5) / (D_R - D_L);
            return t;
        }
    }
}

__global__ void funk_time(double* T, double* T_do, double* TT, int* i)
{
    *T_do = *T;
    *TT = *TT + *T_do;
    *T = 10000000;
    *i = *i + 1;
    if (*i % 5000 == 0)
    {
        printf("i = %d,  TT = %lf \n", *i, *TT);
    }
    return;
}

//__global__ void add(double2* s, double2* u, double2* s2, double2* u2, double* T, double* T_do, int method)
//{
//    int index = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
//    int n = index % N;                                   // ����� ������ �� x (�� 0)
//    int m = (index - n) / N;                             // ����� ������ �� y (�� 0)
//    double y = y_min + m * (y_max) / (M);
//    double x = x_min + n * (x_max - x_min) / (N - 1);
//    double dist = __dsqrt_rn(x * x + y * y);
//
//    // ����� �������������� ������ � �������, ��� ����� ��������� shared ������ � ����������� ������ �� ���������� ������ � ����
//    __shared__ double2 a[THREADS_PER_BLOCK + 2][3];
//    __shared__ double2 b[THREADS_PER_BLOCK + 2][3];
//
//    if (m == 0)
//    {
//        a[threadIdx.x + 1][1] = s[index];
//        a[threadIdx.x + 1][2] = s[(m + 1) * N + n];
//        if ((threadIdx.x == 0) && (n != 0))
//        {
//            a[0][1] = s[index - 1];
//        }
//        if ((threadIdx.x == THREADS_PER_BLOCK - 1) && (n != N - 1))
//        {
//            a[THREADS_PER_BLOCK + 1][1] = s[index + 1];
//        }
//
//        b[threadIdx.x + 1][1] = u[index];
//        b[threadIdx.x + 1][2] = u[(m + 1) * N + n];
//        if ((threadIdx.x == 0) && (n != 0))
//        {
//            b[0][1] = u[index - 1];
//        }
//        if ((threadIdx.x == THREADS_PER_BLOCK - 1) && (n != N - 1))
//        {
//            b[THREADS_PER_BLOCK + 1][1] = u[index + 1];
//        }
//    }
//    else if (m == M - 1)
//    {
//        a[threadIdx.x + 1][1] = s[index];
//        b[threadIdx.x + 1][1] = u[index];
//    }
//    else
//    {
//        a[threadIdx.x + 1][1] = s[index];
//        a[threadIdx.x + 1][0] = s[(m - 1) * N + n];
//        a[threadIdx.x + 1][2] = s[(m + 1) * N + n];
//        if ((threadIdx.x == 0) && (n != 0))
//        {
//            a[0][1] = s[index - 1];
//        }
//        if ((threadIdx.x == THREADS_PER_BLOCK - 1) && (n != N - 1))
//        {
//            a[THREADS_PER_BLOCK + 1][1] = s[index + 1];
//        }
//
//        b[threadIdx.x + 1][1] = u[index];
//        b[threadIdx.x + 1][0] = u[(m - 1) * N + n];
//        b[threadIdx.x + 1][2] = u[(m + 1) * N + n];
//        if ((threadIdx.x == 0) && (n != 0))
//        {
//            b[0][1] = u[index - 1];
//        }
//        if ((threadIdx.x == THREADS_PER_BLOCK - 1) && (n != N - 1))
//        {
//            b[THREADS_PER_BLOCK + 1][1] = u[index + 1];
//        }
//    }
//    // ������� ������.  ����������������
//    __syncthreads();
//
//
//
//    //��������� ������ �����
//    if ((n == N - 1) || (m == M - 1) || (dist < 110)) // Ƹ����� ��������� �������
//    {
//        // � ���� ������� �������� ���������� ������������� � �� �������� � �������� �������)
//        s2[index].x = a[threadIdx.x + 1][1].x;
//        s2[index].y = a[threadIdx.x + 1][1].y;
//        u2[index].x = b[threadIdx.x + 1][1].x;
//        u2[index].y = b[threadIdx.x + 1][1].y;
//        return;
//    }
//
//    double2 s_1, s_2, s_3, s_4, s_5, u_1, u_2, u_3, u_4, u_5;      // ���������� ���� ������� � ����� ������
//    double2 Ps12 = { 0,0 }, Pu12 = { 0,0 }, Ps13 = { 0,0 }, Pu13 = { 0,0 }, //
//        Ps14 = { 0,0 }, Pu14 = { 0,0 }, Ps15 = { 0,0 }, Pu15 = { 0,0 }; // ������� �������
//    double tmin = 1000;
//
//    s_1 = a[threadIdx.x + 1][1];
//    u_1 = b[threadIdx.x + 1][1];
//    s_2 = a[threadIdx.x + 2][1];
//    u_2 = b[threadIdx.x + 2][1];
//    s_3 = a[threadIdx.x + 1][0];
//    u_3 = b[threadIdx.x + 1][0];
//    s_4 = a[threadIdx.x][1];
//    u_4 = b[threadIdx.x][1];
//    s_5 = a[threadIdx.x + 1][2];
//    u_5 = b[threadIdx.x + 1][2];
//
//    if ((n == 0) && (m == 0))
//    {
//        s_3 = s_1;
//        u_3.x = u_1.x;
//        u_3.y = -u_1.y;
//        s_4 = s_1;
//        u_4 = u_1;
//    }
//    else if (n == 0)
//    {
//        s_4 = s_1;
//        u_4 = u_1;
//    }
//    else if (m == 0)
//    {
//        s_3 = s_1;
//        u_3.x = u_1.x;
//        u_3.y = -u_1.y;
//    }
//
//
//    if (method == 0)
//    {
//        tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dy));
//        tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dx));
//        tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dy));
//        tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dx));
//    }
//    else if (method == 1)
//    {
//        tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dy));
//        tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dx));
//        tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dy));
//        tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dx));
//    }
// 
// 
//
//    if (*T > tmin)
//    {
//        __threadfence();
//        *T = tmin;
//    }
//
//    double2 PS = { 0,0 };
//    double2 PU = { 0,0 };
//
//    /*if ((x > 400) &&( x < 405) && (y < 5))
//    {
//        printf("%lf, %lf, %lf, %lf, %lf\n", Ps12.x, Ps13.x, Ps14.x, Ps15.x, Ps12.x* dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx);
//    }*/
//
//    PS.x = Ps12.x * dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx;
//    PS.y = Ps12.y * dy + Ps13.y * dx + Ps14.y * dy + Ps15.y * dx;
//    PU.x = Pu12.x * dy + Pu13.x * dx + Pu14.x * dy + Pu15.x * dx;
//    PU.y = Pu12.y * dy + Pu13.y * dx + Pu14.y * dy + Pu15.y * dx;
//
//    double dV = dx * dy;
//
//    s2[index].x = s[index].x - (*T_do / dV) * PS.x - *T_do * s[index].x * u[index].y / y;
//    if (s2[index].x <= 0)
//    {
//        printf("Problems! x = %lf, y = %lf, ro = %lf\n", x, y, s2[index].x);
//        s2[index].x = 0.000001;
//    }
//    u2[index].x = (s[index].x * u[index].x - (*T_do / dV) * PU.x - *T_do * s[index].x * u[index].y * u[index].x / y) / s2[index].x;
//    u2[index].y = (s[index].x * u[index].y - (*T_do / dV) * PU.y - *T_do * s[index].x * u[index].y * u[index].y / y) / s2[index].x;
//    s2[index].y = (((s[index].y / (ggg - 1) + s[index].x * (u[index].x * u[index].x + u[index].y * u[index].y) * 0.5) - (*T_do / dV) * PS.y - //
//        *T_do * u[index].y * (ggg * s[index].y / (ggg - 1) + s[index].x * (u[index].x * u[index].x + u[index].y * u[index].y) * 0.5) / y) - //
//        0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1);
//    if (s2[index].y <= 0)
//    {
//        s2[index].y = 0.000001;
//    }
//}
//

__global__ void Ker_Dekard(double2* s, double2* u, double2* s2, double2* u2, double* T, double* T_do, int method)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
    int n = index % N;                                   // ����� ������ �� x (�� 0)
    int m = (index - n) / N;                             // ����� ������ �� y (�� 0)
    double y = y_min + m * (y_max) / (M);
    double x = x_min + n * (x_max - x_min) / (N - 1);
    //double dist = __dsqrt_rn(x * x + y * y);

    double2 s_1, s_2, s_3, s_4, s_5, u_1, u_2, u_3, u_4, u_5;      // ���������� ���� ������� � ����� ������
    double2 Ps12 = { 0,0 }, Pu12 = { 0,0 }, Ps13 = { 0,0 }, Pu13 = { 0,0 }, //
        Ps14 = { 0,0 }, Pu14 = { 0,0 }, Ps15 = { 0,0 }, Pu15 = { 0,0 }; // ������� �������
    double tmin = 1000;

    s_1 = s[index];
    u_1 = u[index];
    if ((n == N - 1)||(m == M-1)) // Ƹ����� ��������� �������
    {
        // � ���� ������� �������� ���������� ������������� � �� �������� � �������� �������)
        //s2[index] = s_1;
        //u2[index] = u_1;
        return;
    }
    s_2 = s[(m)*N + n + 1];
    u_2 = u[(m)*N + n + 1];

    if ((n == 0))
    {
        s_4 = s_1;
        u_4 = u_1;
    }
    else
    {
        s_4 = s[(m)*N + n - 1];
        u_4 = u[(m)*N + n - 1];
    }

    if ((m == 0))
    {
        s_3 = s_1;
        u_3 = u_1;
    }
    else
    {
        s_3 = s[(m - 1) * N + (n)];
        u_3 = u[(m - 1) * N + (n)];
    }

    if ((m == M - 1))
    {
        s_5 = s_1;
        u_5 = u_1;
    }
    else
    {
        s_5 = s[(m + 1) * N + (n)];
        u_5 = u[(m + 1) * N + (n)];
    }


    if (method == 0)
    {
        tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dy));
        tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dx));
        tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dy));
        tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dx));
    }
    else if (method == 1)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dy));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dx));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dy));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dx));
        }
    }


    if (*T > tmin)
    {
        //__threadfence();
        *T = tmin;
    }

    double2 PS = { 0,0 };
    double2 PU = { 0,0 };

    /*if ((x > 400) &&( x < 405) && (y < 5))
    {
        printf("%lf, %lf, %lf, %lf, %lf\n", Ps12.x, Ps13.x, Ps14.x, Ps15.x, Ps12.x* dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx);
    }*/

    PS.x = Ps12.x * dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx;
    PS.y = Ps12.y * dy + Ps13.y * dx + Ps14.y * dy + Ps15.y * dx;
    PU.x = Pu12.x * dy + Pu13.x * dx + Pu14.x * dy + Pu15.x * dx;
    PU.y = Pu12.y * dy + Pu13.y * dx + Pu14.y * dy + Pu15.y * dx;

    double dV = dx * dy;

    s2[index].x = s[index].x - (*T_do / dV) * PS.x;
    if (s2[index].x <= 0)
    {
        printf("Problemsssss! x = %lf, y = %lf, ro = %lf\n", x, y, s2[index].x);
        s2[index].x = 0.0001;
    }
    u2[index].x = (s[index].x * u[index].x - (*T_do / dV) * PU.x) / s2[index].x;
    u2[index].y = (s[index].x * u[index].y - (*T_do / dV) * PU.y) / s2[index].x;
    s2[index].y = (((s[index].y / (ggg - 1) + s[index].x * (u[index].x * u[index].x + u[index].y * u[index].y) * 0.5) - (*T_do / dV) * PS.y) - //
        0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1);
    if (s2[index].y <= 0)
    {
        s2[index].y = 0.000001;
    }
}

__global__ void add2(double2* s, double2* u, double2* s2, double2* u2, double* T, double* T_do, int method, int step)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
    int n = index % N;                                   // ����� ������ �� x (�� 0)
    int m = (index - n) / N;                             // ����� ������ �� y (�� 0)
    double y = y_min + m * dy;
    double x = x_min + n * dx;
    double dist = sqrt(x * x + y * y);

    double2 s_1, s_2, s_3, s_4, s_5, u_1, u_2, u_3, u_4, u_5;      // ���������� ���� ������� � ����� ������
    double2 Ps12 = { 0,0 }, Pu12 = { 0,0 }, Ps13 = { 0,0 }, Pu13 = { 0,0 }, //
        Ps14 = { 0,0 }, Pu14 = { 0,0 }, Ps15 = { 0,0 }, Pu15 = { 0,0 }; // ������� �������
    double tmin = 1000;

    if (index < 0 || index > N * M - 1)
    {
        printf("Error index = %d \n", index);
    }

    double n1, n2, nn;

    s_1 = s[index];
    u_1 = u[index];
    double dist2 = kv(x + 0.35) / kv(0.65) + kv(y) / kv(0.55);
    if ( (dist2 < 1.0)  ) // Ƹ����� ��������� �������
    {
        // � ���� ������� �������� ���������� ������������� � �� �������� � �������� �������)
        s2[index] = s_1;
        u2[index] = u_1;
        return;
    }



    if (n == N - 1)
    {
        s_2 = { 1.0, 1.0};
        u_2 = { Velosity_inf, 0.0 };
    }
    else
    {
        s_2 = s[(m)*N + n + 1];
        u_2 = u[(m)*N + n + 1];
    }


    if ((n == 0))
    {
        s_4.x = s_1.x;
        s_4.y = s_1.y;
        u_4 = u_1;
        //u_4.x = -2.2;
        if (  (u_4.x > 0.5 * Velosity_inf)  )
        {
            u_4.x = 0.5 * Velosity_inf;              // ������� ������ ��������
        }
    }
    else
    {
        s_4 = s[(m)*N + n - 1];
        u_4 = u[(m)*N + n - 1];
    }

    if ((m == M - 1))
    {
        s_5 = s_1;
        u_5 = u_1;
    }
    else
    {
        s_5 = s[(m + 1) * N + (n)];
        u_5 = u[(m + 1) * N + (n)];
    }
    
    if ((m == 0))
    {
        s_3 = s_1;
        u_3.x = u_1.x;
        u_3.y = -u_1.y;
    }
    else
    {
        s_3 = s[(m - 1) * N + (n)];
        u_3 = u[(m - 1) * N + (n)];
    }


    if (method == 0)
    {
        tmin = my_min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        tmin = my_min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        tmin = my_min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        tmin = my_min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
    }
    else if (method == 1)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }

    }
    else if (method == 2)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }

    }


    if (*T > tmin)
    {
        //atomicExch(T, tmin);
        *T = tmin;
    }

    double2 PS = { 0.0, 0.0 };
    double2 PU = { 0.0, 0.0 };

    /*if ((x > 400) &&( x < 405) && (y < 5))
    {
        printf("%lf, %lf, %lf, %lf, %lf\n", Ps12.x, Ps13.x, Ps14.x, Ps15.x, Ps12.x* dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx);
    }*/

    PS.x = (Ps12.x + Ps14.x) * dy + (Ps13.x + Ps15.x) * dx;
    PS.y = (Ps12.y + Ps14.y) * dy + (Ps13.y + Ps15.y) * dx;
    PU.x = (Pu12.x + Pu14.x) * dy + (Pu13.x + Pu15.x) * dx;
    PU.y = (Pu12.y + Pu14.y) * dy + (Pu13.y + Pu15.y) * dx;

    double dV = dx * dy;

    s2[index].x = s_1.x - (*T_do / dV) * PS.x - (*T_do/y) * s_1.x * u_1.y;

    //s2[index].x = s_1.x - (*T_do / dV) * PS.x;
    if (s2[index].x <= 0)
    {
        printf("Problemsssss! x = %lf, y = %lf, ro = %lf, T = %lf, ro = %lf \n", x, y, s2[index].x, *T_do, s_1.x);
        s2[index].x = s_1.x;
    }
    u2[index].x = (s_1.x * u_1.x - (*T_do / dV) * PU.x - (*T_do/y) * s_1.x * u_1.y * u_1.x) / s2[index].x;
    u2[index].y = (s_1.x * u_1.y - (*T_do / dV) * PU.y - (*T_do/y) * s_1.x * u_1.y * u_1.y) / s2[index].x;
    s2[index].y = (((s_1.y / (ggg - 1.0) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) - (*T_do / dV) * PS.y - //
        (*T_do/y) * u_1.y * (ggg * s_1.y / (ggg - 1.0) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) ) - //
        0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1.0);
    //u2[index].x = (s_1.x * u_1.x - (*T_do / dV) * PU.x ) / s2[index].x;
    //u2[index].y = (s_1.x * u_1.y - (*T_do / dV) * PU.y) / s2[index].x;
    

    //s2[index].y = ( ( (s_1.y / (ggg - 1) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) - (*T_do / dV) * PS.y ) - //
    //    0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1);
    if (s2[index].y <= 0)
    {
        s2[index].y = 0.000001;
    }

}

__global__ void add_MK(double2* s, double2* u, double2* s2, double2* u2, double* nn1, double3* nn2, double* nn3, //
                            double* T, double* T_do, int method, int step)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
    int n = index % N;                                   // ����� ������ �� x (�� 0)
    int m = (index - n) / N;                             // ����� ������ �� y (�� 0)
    double y = y_min + m * dy;
    double x = x_min + n * dx;
    double dist = sqrt(x * x + y * y);

    double2 s_1, s_2, s_3, s_4, s_5, u_1, u_2, u_3, u_4, u_5;      // ���������� ���� ������� � ����� ������
    double2 Ps12 = { 0,0 }, Pu12 = { 0,0 }, Ps13 = { 0,0 }, Pu13 = { 0,0 }, //
        Ps14 = { 0,0 }, Pu14 = { 0,0 }, Ps15 = { 0,0 }, Pu15 = { 0,0 }; // ������� �������
    double tmin = 1000;

    if (index < 0 || index > N * M - 1)
    {
        printf("Error index = %d \n", index);
    }

    double n1, n2, nn;

    s_1 = s[index];
    u_1 = u[index];
    double dist2 = kv(x + 0.35) / kv(0.65) + kv(y) / kv(0.55);
    if ((dist2 < 1.0)) // Ƹ����� ��������� �������
    {
        // � ���� ������� �������� ���������� ������������� � �� �������� � �������� �������)
        s2[index] = s_1;
        u2[index] = u_1;
        return;
    }



    if (n == N - 1)
    {
        s_2 = { 1.0, 1.0 };
        u_2 = { Velosity_inf, 0.0 };
    }
    else
    {
        s_2 = s[(m)*N + n + 1];
        u_2 = u[(m)*N + n + 1];
    }


    if ((n == 0))
    {
        s_4.x = s_1.x;
        s_4.y = s_1.y;
        u_4 = u_1;
        //u_4.x = -2.2;
        if ((u_4.x > 0.5 * Velosity_inf))
        {
            u_4.x = 0.5 * Velosity_inf;              // ������� ������ ��������
        }
    }
    else
    {
        s_4 = s[(m)*N + n - 1];
        u_4 = u[(m)*N + n - 1];
    }

    if ((m == M - 1))
    {
        s_5 = s_1;
        u_5 = u_1;
    }
    else
    {
        s_5 = s[(m + 1) * N + (n)];
        u_5 = u[(m + 1) * N + (n)];
    }

    if ((m == 0))
    {
        s_3 = s_1;
        u_3.x = u_1.x;
        u_3.y = -u_1.y;
    }
    else
    {
        s_3 = s[(m - 1) * N + (n)];
        u_3 = u[(m - 1) * N + (n)];
    }


    if (method == 0)
    {
        tmin = my_min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        tmin = my_min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        tmin = my_min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        tmin = my_min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
    }
    else if (method == 1)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }

    }
    else if (method == 2)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_2, u_2, 1, 0, Ps12, Pu12, dx));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_3, u_3, 0, -1, Ps13, Pu13, dy));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_4, u_4, -1, 0, Ps14, Pu14, dx));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s_1, u_1, s_5, u_5, 0, 1, Ps15, Pu15, dy));
        }

    }


    if (*T > tmin)
    {
        //atomicExch(T, tmin);
        *T = tmin;
    }

    double2 PS = { 0.0, 0.0 };
    double2 PU = { 0.0, 0.0 };

    /*if ((x > 400) &&( x < 405) && (y < 5))
    {
        printf("%lf, %lf, %lf, %lf, %lf\n", Ps12.x, Ps13.x, Ps14.x, Ps15.x, Ps12.x* dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx);
    }*/

    PS.x = (Ps12.x + Ps14.x) * dy + (Ps13.x + Ps15.x) * dx;
    PS.y = (Ps12.y + Ps14.y) * dy + (Ps13.y + Ps15.y) * dx;
    PU.x = (Pu12.x + Pu14.x) * dy + (Pu13.x + Pu15.x) * dx;
    PU.y = (Pu12.y + Pu14.y) * dy + (Pu13.y + Pu15.y) * dx;

    double dV = dx * dy;

    s2[index].x = s_1.x - (*T_do / dV) * PS.x - (*T_do / y) * s_1.x * u_1.y;

    //s2[index].x = s_1.x - (*T_do / dV) * PS.x;
    if (s2[index].x <= 0)
    {
        printf("Problemsssss! x = %lf, y = %lf, ro = %lf, T = %lf, ro = %lf \n", x, y, s2[index].x, *T_do, s_1.x);
        s2[index].x = s_1.x;
    }
    u2[index].x = (s_1.x * u_1.x - (*T_do / dV) * PU.x - (*T_do / y) * s_1.x * u_1.y * u_1.x + *T_do * (n_H/Kn) * nn2[index].x) / s2[index].x;
    u2[index].y = (s_1.x * u_1.y - (*T_do / dV) * PU.y - (*T_do / y) * s_1.x * u_1.y * u_1.y + *T_do * (n_H / Kn) * nn2[index].y) / s2[index].x;
    s2[index].y = (((s_1.y / (ggg - 1.0) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) - (*T_do / dV) * PS.y - //
        (*T_do / y) * u_1.y * (ggg * s_1.y / (ggg - 1.0) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) + *T_do * (n_H / Kn) * nn3[index]) - //
        0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1.0);
    //u2[index].x = (s_1.x * u_1.x - (*T_do / dV) * PU.x ) / s2[index].x;
    //u2[index].y = (s_1.x * u_1.y - (*T_do / dV) * PU.y) / s2[index].x;


    //s2[index].y = ( ( (s_1.y / (ggg - 1) + s_1.x * (u_1.x * u_1.x + u_1.y * u_1.y) * 0.5) - (*T_do / dV) * PS.y ) - //
    //    0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1);
    if (s2[index].y <= 0)
    {
        s2[index].y = 0.000001;
    }

}

__global__ void Kernel_TVD(double2* s, double2* u, double2* s2, double2* u2, double* T, double* T_do, int method)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
    int n = index % N;                                   // ����� ������ �� x (�� 0)
    int m = (index - n) / N;                             // ����� ������ �� y (�� 0)
    double y = y_min + m * (y_max) / (M);
    double x = x_min + n * (x_max - x_min) / (N - 1);
    double dist = __dsqrt_rn(x * x + y * y);

    double2 s_1, s_2, s_3, s_4, s_5, u_1, u_2, u_3, u_4, u_5;      // ���������� ���� ������� � ����� ������
    double2 Ps12 = { 0,0 }, Pu12 = { 0,0 }, Ps13 = { 0,0 }, Pu13 = { 0,0 }, //
        Ps14 = { 0,0 }, Pu14 = { 0,0 }, Ps15 = { 0,0 }, Pu15 = { 0,0 }; // ������� �������
    double tmin = 1000;

    s_1 = s[index];
    u_1 = u[index];
    //if ((n == N - 1) || (m == M - 1) || (dist < 110)) // Ƹ����� ��������� �������
   if ((n == N - 1) || (dist < 110)) // Ƹ����� ��������� �������
    {
        // � ���� ������� �������� ���������� ������������� � �� �������� � �������� �������)
        s2[index] = s_1;
        u2[index] = u_1;
        return;
    }
    s_2 = s[(m)*N + n + 1];
    u_2 = u[(m)*N + n + 1];

    if (m == M - 1)
    {
        s_5 = s_1;
        u_5 = u_1;
    }
    else
    {
        s_5 = s[(m + 1) * N + n];
        u_5 = u[(m + 1) * N + n];
    }
    if ((n == 0))
    {
        s_4.x = s_1.x;
        s_4.y = 1.0 / (ggg * M_inf * M_inf);      // ������������ ��������
        u_4 = u_1;
        //if (u_1.x > 0.0)
        //{
        //    u_4.x = -0.3;              // ������� ������ ��������
        //}
        double Max = sqrt((u_1.x * u_1.x + u_1.y * u_1.y) / (ggg * s_1.y / s_1.x));
        if ( (u_1.x > -5.0)&&(Max < 1) )
        {
            u_4.x = -5.0;              // ������� �����- ������ ��������
        }
    }
    else
    {
        s_4 = s[(m)*N + n - 1];
        u_4 = u[(m)*N + n - 1];
    }

    if ((m == 0))
    {
        s_3 = s_1;
        u_3.x = u_1.x;
        u_3.y = -u_1.y;
    }
    else
    {
        s_3 = s[(m - 1) * N + (n)];
        u_3 = u[(m - 1) * N + (n)];
    }

    double2 s12 = { 0.0 ,0.0 };
    double2 s13 = { 0.0 ,0.0 };
    double2 s14 = { 0.0 ,0.0 };
    double2 s15 = { 0.0 ,0.0 };
    double2 u12 = { 0.0 ,0.0 };
    double2 u13 = { 0.0 ,0.0 };
    double2 u14 = { 0.0 ,0.0 };
    double2 u15 = { 0.0 ,0.0 };
    double2 s21 = { 0.0 ,0.0 };
    double2 s31 = { 0.0 ,0.0 };
    double2 s41 = { 0.0 ,0.0 };
    double2 s51 = { 0.0 ,0.0 };
    double2 u21 = { 0.0 ,0.0 };
    double2 u31 = { 0.0 ,0.0 };
    double2 u41 = { 0.0 ,0.0 };
    double2 u51 = { 0.0 ,0.0 };
    double A = 0, B = 0;
    // ��������� ���������� �������-�������
    if (n > N - 3)
    {
        s21 = s_2;
        u21 = u_2;
    }
    else
    {
        s21 = s[(m) * N + (n + 2)];
        u21 = u[(m)*N + (n + 2)];
    }
    if (n == 0)
    {
        s41 = s_4;
        u41 = u_4;
    }
    else if (n == 1)
    {
        s41 = s_4;
        u41 = u_4;
        //if (u41.x > 0.0)
        //{
        //    u41.x = -0.3;             // ������� ������ ��������
        //}
        double Max = sqrt((u_4.x * u_4.x + u_4.y * u_4.y) / (ggg * s_4.y / s_4.x));
        if ((u41.x > -5.0) && (Max < 1))
        {
            u41.x = -5.0;              // ������� ������ ��������
        }
    }
    else
    {
        s41 = s[(m)*N + (n - 2)];
        u41 = u[(m)*N + (n - 2)];
    }
    if (m > M - 3)
    {
        s51 = s_5;
        u51 = u_5;
    }
    else
    {
        s51 = s[(m + 2)*N + (n)];
        u51 = u[(m + 2)*N + (n)];
    }
    if (m == 1)
    {
        s31 = s_3;
        u31.x = u_3.x;
        u31.y = -u_3.y;
    }
    else if (m == 0) 
    {
        s31 = s_5;
        u31.x = u_5.x;
        u31.y = -u_5.y;
    }
    else
    {
        s31 = s[(m - 2)*N + (n)];
        u31 = u[(m - 2)*N + (n)];
    }

    linear2(x - dx, s_4.x,      x, s_1.x,   x + dx, s_2.x,  x - dx/2.0, x + dx/2.0,     A, B);
    if (B <= 0)
    {
        s12.x = s_1.x;
    }
    else
    {
        s12.x = B;
    }
    if (A <= 0)
    {
        s14.x = s_1.x;
    }
    else
    {
        s14.x = A;
    }
    linear2(x - dx, s_4.y,       x, s_1.y,    x + dx, s_2.y,  x - dx / 2.0, x + dx / 2.0,    A, B);
    if ((B <= 0) || (grad_p == false) )
    {
        s12.y = s_1.y;
    }
    else
    {
        s12.y = B;
    }
    if ( (A <= 0) || (grad_p == false) )
    {
        s14.y = s_1.y;
    }
    else
    {
        s14.y = A;
    }
    linear2(x - dx, u_4.x,      x, u_1.x,   x + dx, u_2.x,       x - dx / 2.0, x + dx / 2.0,     A, B);
    u12.x = B;
    u14.x = A;
    linear2(x - dx, u_4.y,       x, u_1.y,   x + dx, u_2.y,     x - dx / 2.0, x + dx / 2.0,      A, B);
    u12.y = B;
    u14.y = A;

    linear2(y - dy, s_3.x,      y, s_1.x,       y + dy, s_5.x,      y - dy / 2.0, y + dy / 2.0,     A, B);
    if (B <= 0)
    {
        s15.x = s_1.x;
    }
    else
    {
        s15.x = B;
    }
    if (A <= 0)
    {
        s13.x = s_1.x;
    }
    else
    {
        s13.x = A;
    }
    linear2(y - dy, s_3.y,      y, s_1.y,       y + dy, s_5.y,      y - dy / 2.0, y + dy / 2.0,         A, B);
    if ((B <= 0) || (grad_p == false) )
    {
        s15.y = s_1.y;
    }
    else
    {
        s15.y = B;
    }
    if ( (A <= 0) || (grad_p == false) )
    {
        s13.y = s_1.y;
    }
    else
    {
        s13.y = A;
    }
    linear2(y - dy, u_3.x,      y, u_1.x,       y + dy, u_5.x,       y - dy / 2.0, y + dy / 2.0,        A, B);
    u15.x = B;
    u13.x = A;
    linear2(y - dy, u_3.y,       y, u_1.y,      y + dy, u_5.y,      y - dy / 2.0, y + dy / 2.0,         A, B);
    u15.y = B;
    u13.y = A;

    s21.x = linear(x, s_1.x,     x + dx, s_2.x,      x + 2.0 * dx, s21.x,       x + dx / 2.0);
    if (s21.x <= 0) s21.x = s_2.x;
    s21.y = linear(x, s_1.y,      x + dx, s_2.y,    x + 2.0 * dx, s21.y,    x + dx / 2.0);
    if ( (s21.y <= 0) || (grad_p == false) ) s21.y = s_2.y;
    u21.x = linear(x, u_1.x,    x + dx, u_2.x,      x + 2.0 * dx, u21.x,    x + dx / 2.0);
    u21.y = linear(x, u_1.y,    x + dx, u_2.y,      x + 2.0 * dx, u21.y,    x + dx / 2.0);

    s41.x = linear(x, s_1.x,    x - dx, s_4.x,      x - 2.0 * dx, s41.x,        x - dx / 2.0);
    if (s41.x <= 0) s41.x = s_4.x;
    s41.y = linear(x, s_1.y,    x - dx, s_4.y,      x - 2.0 * dx, s41.y,         x - dx / 2.0);
    if ((s41.y <= 0) || (grad_p == false) ) s41.y = s_4.y;
    u41.x = linear(x, u_1.x,    x - dx, u_4.x,      x - 2.0 * dx, u41.x,        x - dx / 2.0);
    u41.y = linear(x, u_1.y,    x - dx, u_4.y,      x - 2.0 * dx, u41.y,         x - dx / 2.0);

    s31.x = linear(y, s_1.x,        y - dy, s_3.x,      y - 2.0 * dy, s31.x,        y - dy / 2.0);
    if (s31.x <= 0) s31.x = s_3.x;
    s31.y = linear(y, s_1.y,        y - dy, s_3.y,      y - 2.0 * dy, s31.y,        y - dy / 2.0);
    if ( (s31.y <= 0) || (grad_p == false) ) s31.y = s_3.y;
    u31.x = linear(y, u_1.x,        y - dy, u_3.x,      y - 2.0 * dy, u31.x,        y - dy / 2.0);
    u31.y = linear(y, u_1.y,        y - dy, u_3.y,      y - 2.0 * dy, u31.y,        y - dy / 2.0);

    s51.x = linear(y, s_1.x,        y + dy, s_5.x,      y + 2.0 * dy, s51.x,        y + dy / 2.0);
    if (s51.x <= 0) s51.x = s_5.x;
    s51.y = linear(y, s_1.y,        y + dy, s_5.y,      y + 2.0 * dy, s51.y,        y + dy / 2.0);
    if ( (s51.y <= 0)||(grad_p == false) ) s51.y = s_5.y;
    u51.x = linear(y, u_1.x,        y + dy, u_5.x,      y + 2.0 * dy, u51.x,        y + dy / 2.0);
    u51.y = linear(y, u_1.y,        y + dy, u_5.y,      y + 2.0 * dy, u51.y,        y + dy / 2.0);


    if (method == 0)
    {
        tmin = min(tmin, HLL(s12, u12, s21, u21, 1, 0, Ps12, Pu12, dy));
        tmin = min(tmin, HLL(s13, u13, s31, u31, 0, -1, Ps13, Pu13, dx));
        tmin = min(tmin, HLL(s14, u14, s41, u41, -1, 0, Ps14, Pu14, dy));
        tmin = min(tmin, HLL(s15, u15, s51, u51, 0, 1, Ps15, Pu15, dx));
    }
    else if (method == 1)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s12, u12, s21, u21, 1, 0, Ps12, Pu12, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s12, u12, s21, u21, 1, 0, Ps12, Pu12, dy));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s13, u13, s31, u31, 0, -1, Ps13, Pu13, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s13, u13, s31, u31, 0, -1, Ps13, Pu13, dx));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s14, u14, s41, u41, -1, 0, Ps14, Pu14, dy));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s14, u14, s41, u41, -1, 0, Ps14, Pu14, dy));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s15, u15, s51, u51, 0, 1, Ps15, Pu15, dx));
        }
        else
        {
            tmin = min(tmin, HLLC_Aleksashov(s15, u15, s51, u51, 0, 1, Ps15, Pu15, dx));
        }
    }
    else if (method == 2)
    {
        if (x + dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s12, u12, s21, u21, 1, 0, Ps12, Pu12, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s12, u12, s21, u21, 1, 0, Ps12, Pu12, dy));
        }
        if (y - dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s13, u13, s31, u31, 0, -1, Ps13, Pu13, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s13, u13, s31, u31, 0, -1, Ps13, Pu13, dx));
        }
        if (x - dx / 2.0 < hx)
        {
            tmin = min(tmin, HLL(s14, u14, s41, u41, -1, 0, Ps14, Pu14, dy));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s14, u14, s41, u41, -1, 0, Ps14, Pu14, dy));
        }
        if (y + dy / 2.0 < hy)
        {
            tmin = min(tmin, HLL(s15, u15, s51, u51, 0, 1, Ps15, Pu15, dx));
        }
        else
        {
            tmin = min(tmin, Godunov_Solver_Alexashov(s15, u15, s51, u51, 0, 1, Ps15, Pu15, dx));
        }
    }
    else
    {
        printf("Error in method 2375\n");
    }


    if (*T > tmin)
    {
       // __threadfence();
        *T = tmin;
    }

    double2 PS = { 0,0 };
    double2 PU = { 0,0 };

    /*if ((x > 400) &&( x < 405) && (y < 5))
    {
        printf("%lf, %lf, %lf, %lf, %lf\n", Ps12.x, Ps13.x, Ps14.x, Ps15.x, Ps12.x* dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx);
    }*/

    PS.x = Ps12.x * dy + Ps13.x * dx + Ps14.x * dy + Ps15.x * dx;
    PS.y = Ps12.y * dy + Ps13.y * dx + Ps14.y * dy + Ps15.y * dx;
    PU.x = Pu12.x * dy + Pu13.x * dx + Pu14.x * dy + Pu15.x * dx;
    PU.y = Pu12.y * dy + Pu13.y * dx + Pu14.y * dy + Pu15.y * dx;

    double dV = dx * dy;

    s2[index].x = s[index].x - (*T_do / dV) * PS.x - *T_do * s[index].x * u[index].y / y;
    if (s2[index].x <= 0)
    {
        printf("Problemsssss! x = %lf, y = %lf, ro = %lf\n", x, y, s2[index].x);
        s2[index].x = 0.0001;
    }
    u2[index].x = (s[index].x * u[index].x - (*T_do / dV) * PU.x - *T_do * s[index].x * u[index].y * u[index].x / y) / s2[index].x;
    u2[index].y = (s[index].x * u[index].y - (*T_do / dV) * PU.y - *T_do * s[index].x * u[index].y * u[index].y / y) / s2[index].x;
    s2[index].y = (((s[index].y / (ggg - 1) + s[index].x * (u[index].x * u[index].x + u[index].y * u[index].y) * 0.5) - (*T_do / dV) * PS.y - //
        *T_do * u[index].y * (ggg * s[index].y / (ggg - 1) + s[index].x * (u[index].x * u[index].x + u[index].y * u[index].y) * 0.5) / y) - //
        0.5 * s2[index].x * (u2[index].x * u2[index].x + u2[index].y * u2[index].y)) * (ggg - 1);
    if (s2[index].y <= 0)
    {
        s2[index].y = 0.000001;
    }
}

__global__ void test(void)
{
    double2 s_1 = { 1, 0.0666666 };
    double2 u_1 = { -1, 0 };
    double2 s_2 = { 1, 0.0666666 };
    double2 u_2 = { -1, 0 };
    double2 P1, P2;
    Godunov_Solver_Alexashov(s_1, u_1, s_2, u_2, 1, 0, P1, P2, dy);
    printf("%lf\n", P1.x);
    Godunov_Solver_Alexashov(s_1, u_1, s_2, u_2, -1, 0, P1, P2, dy);
    printf("%lf\n", P1.x);
    
}

void print_file_mini(double2* host_s_p, double2* host_u_p, double* nn1, double3* nn2, double* nn3, string name)
{
    ofstream fout;
    fout.open(name);
    int nn = (int)((N + Nmin - 1) / Nmin);
    int mm = (int)((M + Nmin - 1) / Nmin);
    fout << "TITLE = \"HP\"  VARIABLES = \"X\", \"Y\", \"Ro\", \"P\", \"Vx\", \"Vy\", \"Max\", \"T\",\"Ro_H\",\"Vx_H\",\"Vr_H\",\"Vphi_H\",\"T_H\", ZONE T = \"HP\", N = " << nn * mm //
        << " , E = " << (nn - 1) * (mm - 1) << ", F = FEPOINT, ET = quadrilateral" << endl;
    //double ss = (sqv_1 * pi * kv(y_max) + sqv_2 * 2.0 * pi * y_max *  (x_max - x_min + dx));
    for (int k = 0; k < K; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        if ((n % Nmin != 0) || (m % Nmin != 0))
        {
            continue;
        }

        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        double no = (1.0 * AllNumber * (pi * kv(y + dy/2.0) * dx - pi * kv(y - dy / 2.0) * dx));
        double Max = 0.0, Temp = 0.0;
        double nn = nn1[k];// sum_s* nn1[k] / no;
        //double nn = sqv_1 * nn1[k] / no;
        double  n3 = 0.0;
        double v1 = 0.0, v2 = 0.0, v3 = 0.0;
        if (nn1[k] > 0.000001)
        {
            v1 = nn2[k].x / nn1[k];
            v2 = nn2[k].y / nn1[k];
            v3 = nn2[k].z / nn1[k];
            n3 = (2.0 / 3.0) * (nn3[k] / nn1[k] - kvv(v1, v2, v3));
        }

        
        if (host_s_p[k].x > 0.0)
        {
            Max = sqrt((host_u_p[k].x * host_u_p[k].x + host_u_p[k].y * host_u_p[k].y) / (ggg * host_s_p[k].y / host_s_p[k].x));
            Temp = host_s_p[k].y / host_s_p[k].x;
        }
        fout << x << " " << y << " " << host_s_p[k].x << " " << host_s_p[k].y <<//
            " " << host_u_p[k].x << " " << host_u_p[k].y << " " << //
            Max << " " << Temp << " " << nn << " " <<  v1 << " " << v2 << " " << v3 << " " <<  //
            n3 << endl;
    }

    for (int k = 0; k < nn * mm; k = k + 1)
    {
        int n = k % nn;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / nn;
        if ((m < mm - 1) && (n < nn - 1))
        {
            fout << m * nn + n + 1 << " " << m * nn + n + 2 << " " << (m + 1) * nn + n + 2 << " " << (m + 1) * nn + n + 1 << endl;
        }
    }
    fout.close();
}

void Save_file(double2* host_s_p, double2* host_u_p, double* nn1, double3* nn2, double* nn3, string name)
{
    ofstream fout;
    fout.open(name);

    for (int k = 0; k < K; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        fout << x << " " << y << " " << host_s_p[k].x << " " << host_s_p[k].y <<//
            " " << host_u_p[k].x << " " << host_u_p[k].y << " " << nn1[k] << " " << //
            nn2[k].x << " " << nn2[k].y << " " << nn2[k].z << " " << nn3[k] << endl;
    }

    fout.close();
}

int main(void)
{
    double2* host_s, * host_u;
    double2* s, * u;
    double2* host_s2, * host_u2;
    int* host_i;
    double2* s2, * u2;
    int* dev_i;
    double* host_T, * host_T_do, * host_TT;
    double* T, * T_do, * TT;
    int size = K * sizeof(double2);
    double* nn1, * nn3;
    double3* nn2;
    double* dev_nn1, * dev_nn3;
    double3* dev_nn2;

    hipEvent_t start, stop;
    hipError_t cudaStatus;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //�������� ������ ��� device ����� ��� host_s, host_u
    hipMalloc((void**)&s, size);
    hipMalloc((void**)&u, size);
    hipMalloc((void**)&s2, size);
    hipMalloc((void**)&u2, size);
    hipMalloc((void**)&dev_nn1, K * sizeof(double));
    hipMalloc((void**)&dev_nn2, K * sizeof(double3));
    hipMalloc((void**)&dev_nn3, K * sizeof(double));
    hipMalloc((void**)&T, sizeof(double));
    hipMalloc((void**)&T_do, sizeof(double));
    hipMalloc((void**)&TT, sizeof(double));
    hipMalloc((void**)&dev_i, sizeof(int));

    ifstream fin2;
    fin2.open("rnd_Dima.dat");
    vector<Sensor*> Sensors;
    double d, a, b, c;
    for (int i = 0; i < 270; i++)
    {
        fin2 >> d >> a >> b >> c;
        auto s = new Sensor(a, b, c);
        Sensors.push_back(s);
    }

    host_s = (double2*)malloc(size);
    host_u = (double2*)malloc(size);
    host_s2 = (double2*)malloc(size);
    host_u2 = (double2*)malloc(size);
    nn1 = (double*)malloc(K * sizeof(double));
    nn2 = (double3*)malloc(K * sizeof(double3));
    nn3 = (double*)malloc(K * sizeof(double));
    host_T = (double*)malloc(sizeof(double));
    host_T_do = (double*)malloc(sizeof(double));
    host_TT = (double*)malloc(sizeof(double));
    host_i = (int*)malloc(sizeof(int));

    *host_T = 10000000;
    *host_T_do = 0.000000001;
    *host_TT = 0.0;
    *host_i = 0;
    //cout << "dy" << dy << endl;
    //for (int k = 0; k < M; k++)  // ��������� ��������� �������
    //{
    //    double y = y_min + k * (y_max) / (M);
    //    cout << y << endl;
    //}

    double k_ = 0.1;
    double l_ = 1.0;
    double chi = 36.1059; // 36.1059
    std::cout << dx << " " << dy << endl;
    for (int k = 0; k < K; k++)  // ��������� ��������� �������
    {
        nn1[k] = 0.0;
        nn2[k] = { 0.0, 0.0, 0.0 };
        nn3[k] = 0.0;
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * dy;
        double x = x_min + n * dx;
        double dist = sqrt(x * x + y * y);
        double r_0 = 0.00256418;
        double ro = (1.0) / (chi * chi * r_0 * r_0);
        double P_E = ro * chi  * chi / (ggg * 0.2 * 0.2);
        double dist2 = kv(x + 0.35) / kv(0.65) + kv(y) / kv(0.55);
        if (dist2 <= 1.0)
        {
            host_s[k] = { ro * r_0 * r_0/ (dist * dist) , P_E * pow(r_0 / dist, 2.0 * ggg) };
            host_u[k] = { chi * x/dist , chi * y / dist };
            host_s2[k] = { ro * r_0 * r_0 / (dist * dist) , P_E * pow(r_0 / dist, 2.0 * ggg) };
            host_u2[k] = { chi * x / dist , chi * y / dist };
        }
        else 
        {
            host_s[k] = { 1.0, 1.0 };
            host_u[k] = { Velosity_inf, 0.0 };
            host_s2[k] = { 1.0, 1.0 };
            host_u2[k] = { Velosity_inf, 0.0 };
        }
    }


    //
    double c1, c2, a1, a2, a3, a4, a5, a6, a7, a8, a9;
    ifstream fin;
    fin.open("chi_36_start_all.txt"); 

    for (int k = 0; k < K; k++)
    {
        fin >> c1 >> c2 >> a1 >> a2 >> a3 >> a4 >> a5 >> a6 >> a7 >> a8 >> a9;
        host_s[k].x = a1;
        host_s[k].y = a2;
        host_u[k].x = a3;
        host_u[k].y = a4;
        host_s2[k].x = a1;
        host_s2[k].y = a2;
        host_u2[k].x = a3;
        host_u2[k].y = a4;
        nn1[k] = a5;
        nn2[k].x = a6;
        nn2[k].y = a7;
        nn2[k].z = a8;
        nn3[k] = a9;
    }
    fin.close();

    
    for (int k = 0; k < K; k++)  // ��������� ��������� �������
    {
        /*nn1[k] = 0.0;
        nn2[k] = { 0.0, 0.0, 0.0 };
        nn3[k] = 0.0;*/
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * dy;
        double x = x_min + n * dx;
        double dist = sqrt(x * x + y * y);
        double r_0 = 0.00256418;
        double ro = (1.0) / (chi * chi * r_0 * r_0);
        double P_E = ro * chi * chi / (ggg * 0.2 * 0.2);
        double dist2 = kv(x + 0.35) / kv(0.65) + kv(y) / kv(0.55);
        if (dist2 <= 1.0)
        {
            host_s[k] = { ro * r_0 * r_0 / (dist * dist) , P_E * pow(r_0 / dist, 2.0 * ggg) };
            host_u[k] = { chi * x / dist , chi * y / dist };
            host_s2[k] = { ro * r_0 * r_0 / (dist * dist) , P_E * pow(r_0 / dist, 2.0 * ggg) };
            host_u2[k] = { chi * x / dist , chi * y / dist };
        }
    }
    
   
    bool device = true;
    //�������� ���� �� device
    if (device)
    {
        hipMemcpy(s, host_s, size, hipMemcpyHostToDevice);
        hipMemcpy(u, host_u, size, hipMemcpyHostToDevice);
        hipMemcpy(s2, host_s2, size, hipMemcpyHostToDevice);
        hipMemcpy(u2, host_u2, size, hipMemcpyHostToDevice);
        hipMemcpy(T, host_T, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(T_do, host_T_do, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(TT, host_TT, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dev_i, host_i, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_nn1, nn1, K * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(dev_nn2, nn2, K * sizeof(double3), hipMemcpyHostToDevice);
        hipMemcpy(dev_nn2, nn2, K * sizeof(double), hipMemcpyHostToDevice);
    }

    ofstream set;
    set.open("Setka.txt");
    set << "TITLE = \"HP\"  VARIABLES = \"X\", \"Y\"  ZONE T= \"HP\", N="<<  2 * (N + M) << " , E= "<< (N + M) <<", F=FEPOINT, ET=LINESEG" << endl;
    for (int i = 0; i < N; i++)
    {
        double x0 = x_min + i * dx;
        set << x0 - dx/2.0 << " " << 0.0 << endl;
        set << x0 - dx/2.0 << " " << y_max << endl;
    }
    for (int i = 0; i < M; i++)
    {
        double y0 = y_min + i * dy;
        set << x_min - dx/2.0 << " " << y0 - dy/2.0 << endl;
        set << x_max + dx / 2.0 << " " << y0 - dy / 2.0 << endl;
    }
    for (int i = 0; i < N + M; i++)
    {
        set << 2 * i + 1 << " " << 2 * i + 2 << endl;
    }

    std::cout << "START" << endl;


    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "HLL" << endl;
        }
        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 0, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 0, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 50000 == 0 && i > 1)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("8000 step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "HLL" + to_string(i) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);
        }
    }
    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "HLLC" << endl;
        }
        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 1, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 1, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 50000 == 0 || i == 15000 || i == 20000)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("8000 step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "16_09_" + to_string(i) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);
        }
    }
    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "Godunov" << endl;
        }
        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 2, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        add2 << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 2, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 5000 == 0 && i > 1)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("8000 step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "Godunov_" + to_string(i) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);
        }
    }


    for (auto& i: Sensors)
    {
        i->Restart();
    }

    /*M_K(Sensors, host_s, host_u, nn1, nn2, nn3);
    string name = "chi_36_start_all.txt";
    Save_file(host_s, host_u, nn1, nn2, nn3, name);*/

    hipMemcpy(dev_nn1, nn1, K * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_nn2, nn2, K * sizeof(double3), hipMemcpyHostToDevice);
    hipMemcpy(dev_nn2, nn2, K * sizeof(double), hipMemcpyHostToDevice);

    for (int i = 0; i < 400000; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "HLLC_MK" << endl;
        }
        add_MK << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, dev_nn1, dev_nn2, dev_nn3, T, T_do, 1, i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        add_MK << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, dev_nn1, dev_nn2, dev_nn3, T, T_do, 1, i);;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 80000 == 0 || i == 15000 || i == 20000)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "13_03_" + to_string(i) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);
        }
    }

    hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
    hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
    print_file_mini(host_s, host_u, nn1, nn2, nn3, "End_1.txt");
    Save_file(host_s, host_u, nn1, nn2, nn3, "1_plasma_iterate.txt");

 

    if (false)
    {
            // ����� ���������� ���������
        /*Sensor* sens = Sensors[0];
        double Vx1 = 0.0, Vx2 = 0.0;
        double mu1, mu2;
        double ksi3, ksi4, ksi5, ksi6, w;
        double z = 0;
        double p1 = fabs(Velosity_inf) * sqrtpi / (1.0 + fabs(Velosity_inf) * sqrtpi);

        double n1 = 0.0, n2 = 0.0, VV1 = 0.0, T1 = 0.0, n3 = 0.0, n4, VV2, VV3, VV4, T2, T3, T4;*/

        /*ofstream xout;
        xout.open("Vx_do_2.txt");

        ofstream xout2;
        xout2.open("Vx_do_1.txt");

        ofstream xout3;
        xout3.open("Vx_1.txt");

        ofstream xout4;
        xout4.open("Vx_2.txt");*/

        //int kk1 = 800000;
        //int kk2 = 500000;
        //double sq1 = 0.599821;
        //double sq2 = 0.0998206;
        //mu1 = sq1/(sq1 + sq2) * (kk1 + kk2)/kk1;
        //mu2 = sq2 / (sq1 + sq2) * (kk1 + kk2) / kk2;
        //for (int ii = 3; ii < 4; ii++)
        //{
        //    sens = Sensors[ii];
        //    n1 = 0.0;
        //    n2 = 0.0;
        //    n3 = 0.0;
        //    n4 = 0.0;
        //    VV1 = 0.0;
        //    T1 = 0.0;
        //    VV2 = 0.0;
        //    T2 = 0.0;
        //    VV3 = 0.0;
        //    T3 = 0.0;
        //    VV4 = 0.0;
        //    T4 = 0.0;
        //    for (int i = 1; i <= kk1; i++) // ������ � ������ �������
        //    {
        //        z = 0;
        //        Vx1 = 0.0;
        //        Vx2 = 0.0;
        //        p1 = fabs(Velosity_inf) * sqrtpi / (1.0 + fabs(Velosity_inf) * sqrtpi);
        //        do
        //        {
        //            ksi3 = sens->MakeRandom();
        //            ksi4 = sens->MakeRandom();
        //            ksi5 = sens->MakeRandom();
        //            ksi6 = sens->MakeRandom();

        //            if (p1 > ksi3)
        //            {
        //                z = cos(pi * ksi5) * sqrt(-log(ksi4));
        //            }
        //            else
        //            {
        //                if (ksi4 <= 0.5)
        //                {
        //                    z = -sqrt(-log(2.0 * ksi4));
        //                }
        //                else
        //                {
        //                    z = sqrt(-log(2.0 * (1.0 - ksi4)));
        //                }
        //            }
        //        } while (fabs(z + Velosity_inf) / (fabs(Velosity_inf) + fabs(z)) <= ksi6 || z > -Velosity_inf);

        //        Vx1 = z + Velosity_inf;
        //        n1 += mu1 / fabs(Vx1);
        //        VV1 += Vx1 * mu1 / fabs(Vx1);
        //        T1 +=  kv(Vx1) * mu1 / fabs(Vx1);
        //        xout2 << Vx1 << endl;



        //        w = Velosity_inf - Vx1;
        //        p1 = fabs(w) * sqrtpi / (1.0 + fabs(w) * sqrtpi);
        //        do
        //        {
        //            ksi3 = sens->MakeRandom();
        //            ksi4 = sens->MakeRandom();
        //            ksi5 = sens->MakeRandom();
        //            ksi6 = sens->MakeRandom();

        //            if (p1 > ksi3)
        //            {
        //                z = cos(pi * ksi5) * sqrt(-log(ksi4));
        //            }
        //            else
        //            {
        //                if (ksi4 <= 0.5)
        //                {
        //                    z = -sqrt(-log(2.0 * ksi4));
        //                }
        //                else
        //                {
        //                    z = sqrt(-log(2.0 * (1.0 - ksi4)));
        //                }
        //            }
        //        } while (fabs(z + w) / (fabs(w) + fabs(z)) <= ksi6);
        //        Vx2 = z + Velosity_inf;

        //        if (Vx2 > 0)
        //        {
        //            n2 += mu1 / fabs(Vx2);
        //            VV2 += Vx2 * mu1 / fabs(Vx2);
        //            T2 += kv(Vx2) * mu1 / fabs(Vx2);
        //            xout3 << Vx2 << endl;
        //        }
        //        else
        //        {
        //            n4 += mu1 / fabs(Vx2);
        //            VV4 += Vx2 * mu1 / fabs(Vx2);
        //            T4 += kv(Vx2) * mu1 / fabs(Vx2);
        //            xout4 << Vx2 << endl;
        //        }

        //    }
        //    for (int i = 1; i <= kk2; i++)  // ��� ����� �������
        //    {
        //        z = 0;
        //        Vx1 = 0.0;
        //        Vx2 = 0.0;
        //        p1 = fabs(Velosity_inf) * sqrtpi / (1.0 + fabs(Velosity_inf) * sqrtpi);
        //        do
        //        {
        //            ksi3 = sens->MakeRandom();
        //            ksi4 = sens->MakeRandom();
        //            ksi5 = sens->MakeRandom();
        //            ksi6 = sens->MakeRandom();

        //            if (p1 > ksi3)
        //            {
        //                z = cos(pi * ksi5) * sqrt(-log(ksi4));
        //            }
        //            else
        //            {
        //                z = sqrt(-log(1.0 - ksi4));
        //            }
        //        } while (fabs(z + Velosity_inf) / (fabs(Velosity_inf) + fabs(z)) <= ksi6 || z < -Velosity_inf);
        //        Vx1 = z + Velosity_inf;
        //        n3 += mu2 / fabs(Vx1);
        //        VV3 += Vx1 * mu2 / fabs(Vx1);
        //        T3 += mu2 *  kv(Vx1) / fabs(Vx1);
        //        xout << Vx1 << endl;

        //        w = Velosity_inf - Vx1;
        //        p1 = fabs(w) * sqrtpi / (1.0 + fabs(w) * sqrtpi);
        //        do
        //        {
        //            ksi3 = sens->MakeRandom();
        //            ksi4 = sens->MakeRandom();
        //            ksi5 = sens->MakeRandom();
        //            ksi6 = sens->MakeRandom();

        //            if (p1 > ksi3)
        //            {
        //                z = cos(pi * ksi5) * sqrt(-log(ksi4));
        //            }
        //            else
        //            {
        //                if (ksi4 <= 0.5)
        //                {
        //                    z = -sqrt(-log(2.0 * ksi4));
        //                }
        //                else
        //                {
        //                    z = sqrt(-log(2.0 * (1.0 - ksi4)));
        //                }
        //            }
        //        } while (fabs(z + w) / (fabs(w) + fabs(z)) <= ksi6);
        //        Vx2 = z + Velosity_inf;

        //        if (Vx2 > 0)
        //        {
        //            n2 += mu2 / fabs(Vx2);
        //            VV2 += Vx2 * mu2 / fabs(Vx2);
        //            T2 += kv(Vx2) * mu2 / fabs(Vx2);
        //            xout3 << Vx2 << endl;
        //        }
        //        else
        //        {
        //            n4 += mu2 / fabs(Vx2);
        //            VV4 += Vx2 * mu2 / fabs(Vx2);
        //            T4 += kv(Vx2) * mu2 / fabs(Vx2);
        //            xout4 << Vx2 << endl;
        //        }
        //    }

        //    VV1 = VV1 / n1;
        //    T1 = (2.0) * (T1 / n1 - kv(VV1));
        //    n1 = (sq1 + sq2) *  n1 / (kk1 + kk2);

        //    VV2 = VV2 / n2;
        //    T2 = (2.0) * (T2 / n2 - kv(VV2));
        //    n2 = (sq1 + sq2) * n2 / (kk1 + kk2);

        //    VV3 = VV3 / n3;
        //    T3 = (2.0) * (T3 / n3 - kv(VV3));
        //    n3 = (sq1 + sq2) * n3 / (kk1 + kk2);

        //    VV4 = VV4 / n4;
        //    T4 = (2.0) * (T4 / n4 - kv(VV4));
        //    n4 = (sq1 + sq2) * n4 / (kk1 + kk2);

        //    watch(n1);
        //    watch(VV1);
        //    watch(T1);

        //    watch(n2);
        //    watch(VV2);
        //    watch(T2);

        //    watch(n3);
        //    watch(VV3);
        //    watch(T3);

        //    watch(n4);
        //    watch(VV4);
        //    watch(T4);
        //}
    }

    /*M_K(Sensors, host_s, host_u, nn1, nn2, nn3);
    string name = "Godunov_.txt";
    print_file_mini(host_s, host_u, nn1, nn2, nn3, name);*/


    //for (int i = 0; i < 30000; i = i + 2)  // ������� ����� �� ������� ������?
    //{
    //    Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 1);
    //    cudaStatus = hipGetLastError();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //        exit(-1);
    //    }
    //    cudaStatus = hipDeviceSynchronize();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //        exit(-1);
    //    }

    //    funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
    //    cudaStatus = hipGetLastError();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //        exit(-1);
    //    }
    //    cudaStatus = hipDeviceSynchronize();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //        exit(-1);
    //    }

    //    Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 1);
    //    cudaStatus = hipGetLastError();
    //    if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
    //    cudaStatus = hipDeviceSynchronize();
    //    if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

    //    funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
    //    cudaStatus = hipGetLastError();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //        exit(-1);
    //    }
    //    cudaStatus = hipDeviceSynchronize();
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //        exit(-1);
    //    }

    //}

    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "HLL + TVD" << endl;
        }
        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 0);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 0);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 300 == 0 && i > 1)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("300 step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            /*hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "14_06_" + to_string(i+1) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);*/
        }
    }
    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "HLLC + TVD" << endl;
        }
        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 1);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 1);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1); }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 300 == 0 && i > 1)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("300 step - Time:  %.2f sec\n", elapsedTime / 1000.0);
            hipEventRecord(start, 0);
            /*hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "14_06_" + to_string(i+1) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);*/
        }
    }
    for (int i = 0; i < 0; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i == 0)
        {
            cout << "Godunov + tvd" << endl;
        }
        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 2);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "1  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "2  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }
        
        Kernel_TVD << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 2);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "3  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));   exit(-1); }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {fprintf(stderr, "3  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); exit(-1);}
        
        funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            exit(-1);
        }
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "4  hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            exit(-1);
        }

        if (i % 300 == 0 && i>1)
        {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            printf("300 step - Time:  %.2f sec\n", elapsedTime/1000.0);
            hipEventRecord(start, 0);
            /*hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
            hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
            string name = "14_06_" + to_string(i+1) + ".txt";
            print_file_mini(host_s, host_u, nn1, nn2, nn3, name);*/
        }
    }
    //for (int i = 0; i < 20000; i = i + 2)  // ������� ����� �� ������� ������?
    //{
    //    // ��������� add() kernel �� GPU, ��������� ���������
    //    Ker_Dekard << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s, u, s2, u2, T, T_do, 1);
    //    funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
    //    Ker_Dekard << < K / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (s2, u2, s, u, T, T_do, 1);
    //    funk_time << <1, 1 >> > (T, T_do, TT, dev_i);
    //}


    // copy device result back to host copy of c
    if (device)
    {
        hipMemcpy(host_s, s, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_u, u, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_s2, s2, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_u2, u2, size, hipMemcpyDeviceToHost);
        hipMemcpy(host_T, T, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(host_TT, TT, sizeof(double), hipMemcpyDeviceToHost);


        hipEventRecord(stop, 0);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
    }




    printf("Time:  %.2f millisec\n", elapsedTime);

    if (device)
    {
        hipFree(s);
        hipFree(u);
        hipFree(s2);
        hipFree(u2);
        hipFree(T);
        hipFree(T_do);
        hipFree(TT);
        hipFree(dev_i);
        hipFree(dev_nn1);
        hipFree(dev_nn2);
        hipFree(dev_nn3);
    }
    
    ofstream fout;
    fout.open("all_paramets.txt");

    ofstream fout2;
    fout2.open("param_for_texplot.txt");

    ofstream fout5;
    fout5.open("param_for_texplot_mini.txt");

    ofstream fout3;
    fout3.open("param_y=0.txt");

    ofstream fout4;
    fout4.open("inform.txt");

    fout2 << "TITLE = \"HP\"  VARIABLES = \"X\", \"Y\", \"Ro\", \"P\", \"Vx\", \"Vy\", \"Max\", \"T\", ZONE T = \"HP\", N = " << K //
        << " , E = " << (N - 1) * (M - 1) << ", F = FEPOINT, ET = quadrilateral" << endl;
    int nn = (int)((N + Nmin - 1) / Nmin);
    int mm = (int)((M + Nmin - 1) / Nmin);
    fout5 << "TITLE = \"HP\"  VARIABLES = \"X\", \"Y\", \"Ro\", \"P\", \"Vx\", \"Vy\", \"Max\", \"T\", \"Zav\", ZONE T = \"HP\", N = " << nn * mm //
        << " , E = " << (nn - 1)*(mm - 1) << ", F = FEPOINT, ET = quadrilateral" << endl;

    for (int k = 0; k < K; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        fout << x << " " << y << " " << host_s[k].x << " " << host_s[k].y <<//
            " " << host_u[k].x << " " << host_u[k].y << endl;
        //double Max = 0.0, Temp = 0.0;
        //if (host_s[k].x > 0)
        //{
        //    Max = sqrt((host_u[k].x * host_u[k].x + host_u[k].y * host_u[k].y) / (ggg * host_s[k].y / host_s[k].x));
        //    Temp = host_s[k].y / host_s[k].x;
        //}
        //fout2 << x / 184.0 << " " << y / 184.0 << " " << host_s[k].x << " " << host_s[k].y <<//
        //    " " << host_u[k].x << " " << host_u[k].y << " " << //
        //    Max << " " << Temp << endl;
    }

    //for (int k = 0; k < K; k++)
    //{
    //    int n = k % N;                                   // ����� ������ �� x (�� 0)
    //    int m = (k - n) / N;
    //    if ((m < M - 1) && (n < N - 1))
    //    {
    //        fout2 << m * N + n + 1 << " " << m * N + n + 2 << " " << (m + 1) * N + n + 2 << " " << (m + 1) * N + n + 1 << endl;
    //    }
    //}for (int k = 0; k < K; k++)
    //{
    //    int n = k % N;                                   // ����� ������ �� x (�� 0)
    //    int m = (k - n) / N;
    //    if ((m < M - 1) && (n < N - 1))
    //    {
    //        fout2 << m * N + n + 1 << " " << m * N + n + 2 << " " << (m + 1) * N + n + 2 << " " << (m + 1) * N + n + 1 << endl;
    //    }
    //}

    
    for (int k = 0; k < N; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        double ss = 0.0;
        if (host_s[k].x > 0)
        {
            ss = host_s[k].y / pow(host_s[k].x, ggg);
        }
        fout3 << x/184.0 << " " << y/184.0 << " " << host_s[k].x << " " << host_s[k].y <<//
            " " << host_u[k].x << " " << host_u[k].y << " " << ss << endl;
    }
    cout << "TT = " << *host_TT << endl;

    fout4 << "TT = " << *host_TT << "    N = " << N  << "   M = " << M << "   K = " << K  << endl;
    fout4 << "x_min = " << x_min << " " << "x_max = " << x_max << " " << "y_min = " << y_min << " " << "y_max = " << y_max << endl;
    fout4 << "M_inf = " << M_inf << " " << "phi_0 = " << phi_0 << endl;

    int lll = 0;

  

    for (int k = 0; k < K; k++)
    {
        int n = k % N;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / N;                             // ����� ������ �� y (�� 0)
        if ((n % Nmin != 0) || (m % Nmin != 0))
        {
            continue;
        }
        lll++;

        double zav = 0.0;
        if (n > 0 && m > 0 && n < N - 1 && m < M - 1)
        {
            zav = (host_u[(m)*N + n + 1].y - host_u[(m)*N + n - 1].y) / (2 * dx) - (host_u[(m + 1) * N + n].x - host_u[(m - 1) * N + n].x) / (2 * dy);
        }

        double y = y_min + m * (y_max) / (M);
        double x = x_min + n * (x_max - x_min) / (N - 1);
        double Max = 0.0, Temp = 0.0;
        if (host_s[k].x > 0.0)
        {
            Max = sqrt((host_u[k].x * host_u[k].x + host_u[k].y * host_u[k].y) / (ggg * host_s[k].y / host_s[k].x));
            Temp = host_s[k].y / host_s[k].x;
        }
        fout5 << x/184.0 << " " << y/184.0 << " " << host_s[k].x << " " << host_s[k].y <<//
            " " << host_u[k].x << " " << host_u[k].y << " " << //
            Max << " " << Temp << " "  << zav << endl;
    }
    cout << lll << " = lll " << endl;
    cout << nn << " = nn " << endl;
    cout << mm << " = mm " << endl;

    for (int k = 0; k < nn * mm; k = k + 1)
    {
        int n = k % nn;                                   // ����� ������ �� x (�� 0)
        int m = (k - n) / nn;
        if ((m < mm - 1) && (n < nn - 1))
        {
            fout5 << m * nn + n + 1 << " " << m * nn + n + 2 << " " << (m + 1) * nn + n + 2 << " " << (m + 1) * nn + n + 1 << endl;
        }
    }

    fout.close();
    fout2.close();
    fout3.close();
    fout4.close();
    fout5.close();

    return 0;
}